// ----------------------------------------------------------------
// Gunrock -- Fast and Efficient GPU Graph Library
// ----------------------------------------------------------------
// This source code is distributed under the terms of LICENSE.TXT
// in the root directory of this source distribution.
// ----------------------------------------------------------------

/**
 * @file
 * test_utils.cu
 *
 * @brief Utility Routines for Tests
 */
#include <gunrock/util/test_utils.h>

namespace gunrock {
namespace util {

/******************************************************************************
 * Device initialization
 ******************************************************************************/

void DeviceInit(CommandLineArgs &args) {
  int deviceCount;
  hipGetDeviceCount(&deviceCount);
  if (deviceCount == 0) {
    fprintf(stderr, "No devices supporting CUDA.\n");
    exit(1);
  }
  int dev = 0;
  args.GetCmdLineArgument("device", dev);
  if (dev < 0) {
    dev = 0;
  }
  if (dev > deviceCount - 1) {
    dev = deviceCount - 1;
  }
  hipDeviceProp_t deviceProp;
  hipGetDeviceProperties(&deviceProp, dev);
  if (deviceProp.major < 1) {
    fprintf(stderr, "Device does not support CUDA.\n");
    exit(1);
  }
  if (!args.CheckCmdLineFlag("quiet")) {
    printf("Using device %d: %s\n", dev, deviceProp.name);
  }

  hipSetDevice(dev);
}

}  // util
}  // gunrock
