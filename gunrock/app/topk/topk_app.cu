// ----------------------------------------------------------------------------
// Gunrock -- Fast and Efficient GPU Graph Library
// ----------------------------------------------------------------------------
// This source code is distributed under the terms of LICENSE.TXT
// in the root directory of this source distribution.
// ----------------------------------------------------------------------------

/**
 * @file topk_app.cu
 *
 * @brief top k degree centralities application
 */

#include <gunrock/gunrock.h>
#include <gunrock/graphio/market.cuh>
#include <gunrock/app/topk/topk_enactor.cuh>
#include <gunrock/app/topk/topk_problem.cuh>

using namespace gunrock;
using namespace gunrock::util;
using namespace gunrock::oprtr;
using namespace gunrock::app::topk;

/*
 * @brief searches for a value in sorted array
 *
 * @tparam VertexId
 * @tparam SizeT
 *
 * @param[in] arr is an array to search in
 * @param[in] val is searched value
 * @param[in] left  is an index of left  boundary
 * @param[in] right is an index of right boundary
 *
 * return the searched value, if it presents in the array
 * return -1 if the searched value is absent
 */
template <typename VertexId, typename SizeT>
int binary_search(VertexId *arr, VertexId val, SizeT left, SizeT right) {
  while (left <= right) {
    int mid = left + (right - left) / 2;
    if (arr[mid] == val) {
      return arr[mid];
    } else if (arr[mid] > val) {
      right = mid - 1;
    } else {
      left = mid + 1;
    }
  }
  return -1;
}

/**
 * @brief Build Sub-Graph Contains Only Top K Nodes
 *
 * @tparam VertexId
 * @tparam SizeT
 *
 * @param[out] subgraph output subgraph of topk problem
 * @param[in]  graph_original input graph need to process on
 * @param[in]  graph_reversed reversed input graph need to process on
 * @param[out] node_ids output top-k node_ids
 * @param[in] top_nodes Number of nodes needed to process on
 */
template <typename VertexId, typename Value, typename SizeT>
void build_topk_subgraph(GRGraph *subgraph,
                         const Csr<VertexId, Value, SizeT> &graph_original,
                         const Csr<VertexId, Value, SizeT> &graph_reversed,
                         VertexId *node_ids, int top_nodes) {
  int search_return = 0;
  int search_count = 0;
  std::vector<VertexId> node_ids_vec(node_ids, node_ids + top_nodes);
  std::vector<int> sub_row_offsets;
  std::vector<VertexId> sub_col_indices;

  // build row_offsets and col_indices of sub-graph
  sub_row_offsets.push_back(0);  // start of row_offsets
  for (int i = 0; i < top_nodes; ++i) {
    for (int j = 0; j < top_nodes; ++j) {
      /*
      // debug print
      printf("searching %d in column_indices[%d, %d) = [", node_ids[j],
      graph_original.row_offsets[node_ids[i]],
      graph_original.row_offsets[node_ids[i]+1]);
      for (int k = graph_original.row_offsets[node_ids[i]];
      k < graph_original.row_offsets[node_ids[i]+1]; ++k)
      {
      printf(" %d", graph_original.column_indices[k]);
      }
      printf("]\n");
      */
      search_return = binary_search<VertexId, SizeT>(
          graph_original.column_indices, node_ids[j],
          graph_original.row_offsets[node_ids[i]],       // [left
          graph_original.row_offsets[node_ids[i] + 1]);  // right)
      // filter col_indices
      if (search_return != -1) {
        ++search_count;
        // TODO: improve efficiency
        search_return =
            std::find(node_ids_vec.begin(), node_ids_vec.end(), search_return) -
            node_ids_vec.begin();
        sub_col_indices.push_back(search_return);
      }
    }
    // build sub_row_offsets
    search_count += sub_row_offsets[sub_row_offsets.size() - 1];
    sub_row_offsets.push_back(search_count);
    search_count = 0;
  }

  // generate subgraph of top k nodes
  subgraph->num_nodes = top_nodes;
  subgraph->num_edges = sub_col_indices.size();
  subgraph->row_offsets = &sub_row_offsets[0];
  subgraph->col_indices = &sub_col_indices[0];

  /*
  // display sub-graph
  Csr<int, int, int> test_graph(false);
  test_graph.nodes = subgraph->num_nodes;
  test_graph.edges = subgraph->num_edges;
  test_graph.row_offsets    = (int*)subgraph->row_offsets;
  test_graph.column_indices = (int*)subgraph->col_indices;

  test_graph.DisplayGraph();

  test_graph.row_offsets    = NULL;
  test_graph.column_indices = NULL;
  */

  // clean up
  node_ids_vec.clear();
  sub_row_offsets.clear();
  sub_col_indices.clear();
}

/**
 * @brief Run TopK
 *
 * @tparam VertexId
 * @tparam Value
 * @tparam SizeT
 *
 * @param[out] graph_out output subgraph of topk problem
 * @param[out] node_ids return the top k nodes
 * @param[out] in_degrees  return associated centrality
 * @param[out] out_degrees return associated centrality
 * @param[in]  graph_original graph to the CSR graph we process on
 * @param[in]  graph_reversed graph to the CSR graph we process on
 * @param[in]  top_nodes k value for topk problem
 *
 */
template <typename VertexId, typename Value, typename SizeT>
void run_topk(GRGraph *graph_out, VertexId *node_ids, Value *in_degrees,
              Value *out_degrees,
              const Csr<VertexId, Value, SizeT> &graph_original,
              const Csr<VertexId, Value, SizeT> &graph_reversed,
              SizeT top_nodes) {
  typedef TOPKProblem<VertexId, SizeT, Value> Problem;
  TOPKEnactor<false> enactor(false);
  Problem *problem = new Problem;
  top_nodes =
      (top_nodes > graph_original.nodes) ? graph_original.nodes : top_nodes;

  util::GRError(problem->Init(false, graph_original, graph_reversed, 1),
                "Problem TOPK Initialization Failed", __FILE__, __LINE__);

  util::GRError(problem->Reset(enactor.GetFrontierType()),
                "TOPK Problem Data Reset Failed", __FILE__, __LINE__);

  util::GRError(enactor.template Enact<Problem>(problem, top_nodes),
                "TOPK Problem Enact Failed", __FILE__, __LINE__);

  util::GRError(problem->Extract(node_ids, in_degrees, out_degrees, top_nodes),
                "TOPK Problem Data Extraction Failed", __FILE__, __LINE__);

  // build vertex-induced subgraph contains only top k nodes
  build_topk_subgraph<VertexId, Value, SizeT>(
      graph_out, graph_original, graph_reversed, (int *)node_ids, top_nodes);

  if (problem) {
    delete problem;
  }
  hipDeviceSynchronize();
}

/**
 * @brief dispatch function to handle data_types
 *
 * @param[out] graph_o     GRGraph type output
 * @param[out] node_ids    output top k node ids
 * @param[out] in_degrees  output top k in-degree centralities
 * @param[out] out_degrees output top k out-degree centralities
 * @param[in]  graph_i     GRGraph type input graph
 * @param[in]  config      topk specific configurations
 * @param[in]  data_t      topk data_t configurations
 */
void dispatch_topk(GRGraph *graph_o, void *node_ids, void *in_degrees,
                   void *out_degrees, const GRGraph *graph_i,
                   const GRSetup config, const GRTypes data_t) {
  switch (data_t.VTXID_TYPE) {
    case VTXID_INT: {
      switch (data_t.SIZET_TYPE) {
        case SIZET_INT: {
          switch (data_t.VALUE_TYPE) {
            case VALUE_INT: {  // template type = <int, int, int>
              Csr<int, int, int> graph_original(false);
              graph_original.nodes = graph_i->num_nodes;
              graph_original.edges = graph_i->num_edges;
              graph_original.row_offsets = (int *)graph_i->row_offsets;
              graph_original.column_indices = (int *)graph_i->col_indices;
              Csr<int, int, int> graph_reversed(false);
              graph_reversed.nodes = graph_i->num_nodes;
              graph_reversed.edges = graph_i->num_edges;
              graph_reversed.row_offsets = (int *)graph_i->col_offsets;
              graph_reversed.column_indices = (int *)graph_i->row_indices;

              run_topk<int, int, int>(graph_o, (int *)node_ids,
                                      (int *)in_degrees, (int *)out_degrees,
                                      graph_original, graph_reversed,
                                      config.top_nodes);

              // reset for free memory
              graph_original.row_offsets = NULL;
              graph_original.column_indices = NULL;
              graph_reversed.row_offsets = NULL;
              graph_reversed.column_indices = NULL;
              break;
            }
            case VALUE_UINT: {  // template type = <int, uint, int>
              printf("Not Yet Support This DataType Combination.\n");
              break;
            }
            case VALUE_FLOAT: {  // template type = <int, float, int>
              printf("Not Yet Support This DataType Combination.\n");
              break;
            }
          }
          break;
        }
      }
      break;
    }
  }
}

/*
 * @brief topk dispatch function base on gunrock data types
 *
 * @param[out] graph_o     output subgraph of topk problem
 * @param[out] node_ids    output top k node_ids
 * @param[out] in_degrees  output associated centrality values
 * @param[out] out_degrees output associated centrality values
 * @param[in]  graph_i     input graph need to process on
 * @param[in]  config      gunrock primitive specific configurations
 * @param[in]  data_t      gunrock data_t struct
 */
void gunrock_topk(GRGraph *graph_o, void *node_ids, void *in_degrees,
                  void *out_degrees, const GRGraph *graph_i,
                  const GRSetup config, const GRTypes data_t) {
  dispatch_topk(graph_o, node_ids, in_degrees, out_degrees, graph_i, config,
                data_t);
}

// Leave this at the end of the file
// Local Variables:
// mode:c++
// c-file-style: "NVIDIA"
// End:
