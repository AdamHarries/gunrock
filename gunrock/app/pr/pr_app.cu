// ----------------------------------------------------------------------------
// Gunrock -- Fast and Efficient GPU Graph Library
// ----------------------------------------------------------------------------
// This source code is distributed under the terms of LICENSE.TXT
// in the root directory of this source distribution.
// ----------------------------------------------------------------------------

/**
 * @file pr_app.cu
 *
 * @brief Gunrock PageRank Implementation
 */

#include <stdio.h>
#include <gunrock/gunrock.h>

// Graph construction utils
#include <gunrock/graphio/market.cuh>

// Page Rank includes
#include <gunrock/app/pr/pr_enactor.cuh>
#include <gunrock/app/pr/pr_problem.cuh>
#include <gunrock/app/pr/pr_functor.cuh>

// Moderngpu include
#include <moderngpu.cuh>

using namespace gunrock;
using namespace gunrock::util;
using namespace gunrock::oprtr;
using namespace gunrock::app::pr;

/**
 * @brief run page rank
 *
 * @tparam VertexId
 * @tparam Value
 * @tparam SizeT
 *
 * @param[out] ggraph_out Pointer to output CSR graph
 * @param[out] node_ids Pointer to output node IDs
 * @param[out] page_rank Pointer to output PageRanks
 * @param[in] graph Reference to the CSR graph we process on
 * @param[in] source Source ID for personalized PageRank (-1 for general PageRank)
 * @param[in] delta Delta value for computing Page Rank, usually set to .85
 * @param[in] error Error threshold value
 * @param[in] max_iter Max iteration for Page Rank computing
 * @param[in] max_grid_size Maximum CTA occupancy
 * @param[in] num_gpus Number of GPUs
 * @param[in] context CudaContext for moderngpu to use
 */
template <
    typename VertexId,
    typename Value,
    typename SizeT >
void run_page_rank(
    GunrockGraph   *ggraph_out,
    VertexId       *node_ids,
    Value          *page_rank,
    const Csr<VertexId, Value, SizeT> &graph,
    const VertexId source,
    const Value    delta,
    const Value    error,
    const SizeT    max_iter,
    const int      max_grid_size,
    const int      num_gpus,
    CudaContext&   context) {
    typedef PRProblem <
        VertexId,
        SizeT,
        Value > Problem;

    // Allocate host-side label array for gpu-computed results
    //Value    *h_rank    = (Value*)malloc(sizeof(Value) * graph.nodes);
    //VertexId *h_node_id = (VertexId*)malloc(sizeof(VertexId) * graph.nodes);

    // Allocate Page Rank enactor map
    PREnactor<false> pr_enactor(false);

    // Allocate problem on GPU
    Problem *csr_problem = new Problem;
    util::GRError(csr_problem->Init(
                      false,
                      graph,
                      num_gpus),
                  "PageRank Problem Initialization Failed", __FILE__, __LINE__);

    // Perform PageRank
    GpuTimer gpu_timer;

    util::GRError(csr_problem->Reset(
                      source, delta, error, pr_enactor.GetFrontierType()),
                  "PageRank Problem Data Reset Failed", __FILE__, __LINE__);
    gpu_timer.Start();
    util::GRError(pr_enactor.template Enact<Problem>(
                      context, csr_problem, max_iter, max_grid_size),
                  "PageRank Problem Enact Failed", __FILE__, __LINE__);
    gpu_timer.Stop();

    float elapsed = gpu_timer.ElapsedMillis();

    // Copy out results
    util::GRError(csr_problem->Extract(page_rank, node_ids),
                  "PageRank Problem Data Extraction Failed",
                  __FILE__, __LINE__);

    // Cleanup
    if (csr_problem) delete csr_problem;
    //if (h_node_id)   free(h_node_id);
    //if (h_rank)      free(h_rank);

    hipDeviceSynchronize();
}

/**
 * @brief dispatch function to handle data_types
 *
 * @param[out] ggraph_out output of pr problem
 * @param[out] node_ids   output of pr problem
 * @param[out] page_rank  output of pr problem
 * @param[in]  ggraph_in  GunrockGraph type input graph
 * @param[in]  pr_config  pr specific configurations
 * @param[in]  data_type  data type configurations
 * @param[in]  context    moderngpu context
 */
void dispatch_page_rank(
    GunrockGraph          *ggraph_out,
    void                  *node_ids,
    void                  *page_rank,
    const GunrockGraph    *ggraph_in,
    const GunrockConfig   pr_config,
    const GunrockDataType data_type,
    CudaContext&          context) {
    switch (data_type.VTXID_TYPE) {
    case VTXID_INT: {
        switch (data_type.SIZET_TYPE) {
        case SIZET_INT: {
            switch (data_type.VALUE_TYPE) {
            case VALUE_INT: {
                // template type = <int, int, int>
                printf("Not Yet Support This DataType Combination.\n");
                break;
            }
            case VALUE_UINT: {
                // template type = <int, uint, int>
                printf("Not Yet Support This DataType Combination.\n");
                break;
            }
            case VALUE_FLOAT: {
                // template type = <int, float, int>
                // build input csr format graph
                Csr<int, float, int> csr_graph(false);
                csr_graph.nodes          = ggraph_in->num_nodes;
                csr_graph.edges          = ggraph_in->num_edges;
                csr_graph.row_offsets    = (int*)ggraph_in->row_offsets;
                csr_graph.column_indices = (int*)ggraph_in->col_indices;

                // page rank configurations
                float delta         = 0.85f; //!< default delta value
                float error         = 0.01f; //!< error threshold
                int   max_iter      = 20;    //!< maximum number of iterations
                int   max_grid_size = 0;     //!< 0: leave it up to the enactor
                int   num_gpus      = 1;     //!< for multi-gpu enactor to use
                int   src_node      = -1;    //!< source node to start

                // determine source vertex to start sssp
                switch (pr_config.src_mode) {
                case randomize: {
                    src_node = graphio::RandomNode(csr_graph.nodes);
                    break;
                }
                case largest_degree: {
                    int max_node = 0;
                    src_node = csr_graph.GetNodeWithHighestDegree(max_node);
                    break;
                }
                case manually: {
                    src_node = pr_config.src_node;
                    break;
                }
                default: {
                    src_node = -1;
                    break;
                }
                }
                delta    = pr_config.delta;
                error    = pr_config.error;
                max_iter = pr_config.max_iter;

                run_page_rank<int, float, int>(
                    ggraph_out,
                    (int*)node_ids,
                    (float*)page_rank,
                    csr_graph,
                    src_node,
                    delta,
                    error,
                    max_iter,
                    max_grid_size,
                    num_gpus,
                    context);

                // reset for free memory
                csr_graph.row_offsets    = NULL;
                csr_graph.column_indices = NULL;
                break;
            }
            }
            break;
        }
        }
        break;
    }
    }
}

/**
 * @brief run_page_rank entry
 *
 * @param[out] ggraph_out output of pr problem
 * @param[out] node_ids   output of pr problem
 * @param[out] page_rank  output of pr problem
 * @param[in]  ggraph_in  input graph need to process on
 * @param[in]  pr_config  gunrock primitive specific configurations
 * @param[in]  data_type  gunrock datatype struct
 */
void gunrock_pr_func(
    GunrockGraph          *ggraph_out,
    void                  *node_ids,
    void                  *page_rank,
    const GunrockGraph    *ggraph_in,
    const GunrockConfig   pr_config,
    const GunrockDataType data_type) {

    // moderngpu preparations
    int device = 0;
    device = pr_config.device;
    ContextPtr context = mgpu::CreateCudaDevice(device);

    // luanch dispatch function
    dispatch_page_rank(
        ggraph_out,
        node_ids,
        page_rank,
        ggraph_in,
        pr_config,
        data_type,
        *context);
}

// Leave this at the end of the file
// Local Variables:
// mode:c++
// c-file-style: "NVIDIA"
// End:
