// ----------------------------------------------------------------------------
// Gunrock -- Fast and Efficient GPU Graph Library
// ----------------------------------------------------------------------------
// This source code is distributed under the terms of LICENSE.TXT
// in the root directory of this source distribution.
// ----------------------------------------------------------------------------

/**
 * @file cc_app.cu
 *
 * @brief connected component (CC) application
 */

#include <gunrock/gunrock.h>

// graph construction utilities
#include <gunrock/graphio/market.cuh>

// connected component includes
#include <gunrock/app/cc/cc_enactor.cuh>
#include <gunrock/app/cc/cc_problem.cuh>
#include <gunrock/app/cc/cc_functor.cuh>

using namespace gunrock;
using namespace gunrock::util;
using namespace gunrock::oprtr;
using namespace gunrock::app::cc;

/**
 * @brief Run tests for connected component algorithm
 *
 * @tparam VertexId
 * @tparam Value
 * @tparam SizeT
 *
 * @param[out] graph_o Pointer to output CSR graph
 * @param[in] csr_graph Reference to the CSR graph we process on
 * @param[in] max_grid_size Maximum CTA occupancy for CC kernels
 * @param[in] num_gpus Number of GPUs
 */
template <typename VertexId, typename Value, typename SizeT>
void run_cc(GRGraph* graph_o, unsigned int* components,
            const Csr<VertexId, Value, SizeT>& csr, const int max_grid_size,
            const int num_gpus) {
  typedef CCProblem<VertexId, SizeT, Value, true> Problem;  // double buffer

  // Allocate host-side label array for gpu-computed results
  VertexId* h_component_ids = (VertexId*)malloc(sizeof(VertexId) * csr.nodes);
  CCEnactor<false> cc_enactor(false);  // Allocate CC enactor map
  Problem* problem = new Problem;      // Allocate problem on GPU

  util::GRError(problem->Init(false, csr, num_gpus),
                "CC Problem Initialization Failed", __FILE__, __LINE__);

  util::GRError(problem->Reset(cc_enactor.GetFrontierType()),
                "CC Problem Data Reset Failed", __FILE__, __LINE__);

  GpuTimer gpu_timer;
  float elapsed = 0.0f;
  gpu_timer.Start();  // start

  util::GRError(cc_enactor.template Enact<Problem>(problem, max_grid_size),
                "CC Problem Enact Failed", __FILE__, __LINE__);

  gpu_timer.Stop();
  elapsed = gpu_timer.ElapsedMillis();  // elapsed time
  printf(" device elapsed time: %.4f ms\n", elapsed);

  util::GRError(problem->Extract(h_component_ids),
                "CC Problem Data Extraction Failed", __FILE__, __LINE__);

  // Compute number of components in graph
  unsigned int temp = problem->num_components;
  *components = temp;

  // copy component_id per node to GRGraph struct
  graph_o->node_values = (int*)&h_component_ids[0];

  if (problem) delete problem;
  hipDeviceSynchronize();
}

/**
 * @brief dispatch function to handle data_types
 *
 * @param[out] graph_o GRGraph type output
 * @param[in]  graph_i GRGraph type input graph
 * @param[in]  config  cc specific configurations
 * @param[in]  data_t  data type configurations
 */
void dispatch_cc(GRGraph* graph_o, unsigned int* components,
                 const GRGraph* graph_i, const GRSetup config,
                 const GRTypes data_t) {
  switch (data_t.VTXID_TYPE) {
    case VTXID_INT: {
      switch (data_t.SIZET_TYPE) {
        case SIZET_INT: {
          switch (data_t.VALUE_TYPE) {
            case VALUE_INT: {  // template type = <int, int, int>
              // build input csr format graph
              Csr<int, int, int> csr_graph(false);
              csr_graph.nodes = graph_i->num_nodes;
              csr_graph.edges = graph_i->num_edges;
              csr_graph.row_offsets = (int*)graph_i->row_offsets;
              csr_graph.column_indices = (int*)graph_i->col_indices;

              int max_grid_size = 0;  // 0: leave it up to the enactor
              int num_gpus = 1;       // number of GPUs

              run_cc<int, int, int>(graph_o, (unsigned int*)components,
                                    csr_graph, max_grid_size, num_gpus);

              // reset for free memory
              csr_graph.row_offsets = NULL;
              csr_graph.column_indices = NULL;
              break;
            }
            case VALUE_UINT: {  // template type = <int, uint, int>
              printf("Not Yet Support This DataType Combination.\n");
              break;
            }
            case VALUE_FLOAT: {  // template type = <int, float, int>
              printf("Not Yet Support This DataType Combination.\n");
              break;
            }
          }
          break;
        }
      }
      break;
    }
  }
}

/*
 * @brief gunrock_cc function
 *
 * @param[out] graph_o output subgraph of cc problem
 * @param[in]  graph_i input graph need to process on
 * @param[in]  config  primitive specific configurations
 * @param[in]  data_t  gunrock data_t struct
 */
void gunrock_cc(GRGraph* graph_o, unsigned int* components,
                const GRGraph* graph_i, const GRSetup config,
                const GRTypes data_t) {
  dispatch_cc(graph_o, components, graph_i, config, data_t);
}

/*
 * @brief Simple interface take in CSR arrays as input
 * @param[out] components  Return component ID for each node
 * @param[out] num_comps   Return number of components calculated
 * @param[in]  num_nodes   Number of nodes of the input graph
 * @param[in]  num_edges   Number of edges of the input graph
 * @param[in]  row_offsets CSR-formatted graph input row offsets
 * @param[in]  col_indices CSR-formatted graph input column indices
 */
int cc(int* components, const int num_nodes, const int num_edges,
       const int* row_offsets, const int* col_indices) {
  printf("-------------------- setting --------------------\n");

  struct GRTypes data_t;          // primitive-specific data types
  data_t.VTXID_TYPE = VTXID_INT;  // integer
  data_t.SIZET_TYPE = SIZET_INT;  // integer
  data_t.VALUE_TYPE = VALUE_INT;  // integer

  struct GRSetup config;  // primitive-specific configures
  config.device = 0;      // setting device to run

  unsigned int num_components = 0;
  struct GRGraph* graph_o = (struct GRGraph*)malloc(sizeof(struct GRGraph));
  struct GRGraph* graph_i = (struct GRGraph*)malloc(sizeof(struct GRGraph));

  graph_i->num_nodes = num_nodes;
  graph_i->num_edges = num_edges;
  graph_i->row_offsets = (void*)&row_offsets[0];
  graph_i->col_indices = (void*)&col_indices[0];

  printf(" loaded %d nodes and %d edges\n", num_nodes, num_edges);

  printf("-------------------- running --------------------\n");
  gunrock_cc(graph_o, &num_components, graph_i, config, data_t);
  memcpy(components, (int*)graph_o->node_values, num_nodes * sizeof(int));

  if (graph_i) free(graph_i);
  if (graph_o) free(graph_o);

  printf("------------------- completed -------------------\n");
  return num_components;
}

// Leave this at the end of the file
// Local Variables:
// mode:c++
// c-file-style: "NVIDIA"
// End:
