// ----------------------------------------------------------------------------
// Gunrock -- Fast and Efficient GPU Graph Library
// ----------------------------------------------------------------------------
// This source code is distributed under the terms of LICENSE.TXT
// in the root directory of this source distribution.
// ----------------------------------------------------------------------------

/**
 * @file cc_app.cu
 *
 * @brief connected component implementation.
 */

#include <stdio.h>
#include <string>
#include <deque>
#include <vector>
#include <iostream>
#include <gunrock/gunrock.h>

// Graph construction utils
#include <gunrock/graphio/market.cuh>

// CC includes
#include <gunrock/app/cc/cc_enactor.cuh>
#include <gunrock/app/cc/cc_problem.cuh>
#include <gunrock/app/cc/cc_functor.cuh>

using namespace gunrock;
using namespace gunrock::util;
using namespace gunrock::oprtr;
using namespace gunrock::app::cc;

/**
 * @brief Run tests for connected component algorithm
 *
 * @tparam VertexId
 * @tparam Value
 * @tparam SizeT
 *
 * @param[out] ggraph_out Pointer to output CSR graph
 * @param[in] csr_graph Reference to the CSR graph we process on
 * @param[in] max_grid_size Maximum CTA occupancy for CC kernels
 * @param[in] num_gpus Number of GPUs
 */
template <
    typename VertexId,
    typename Value,
    typename SizeT >
void run_cc(
    GunrockGraph *ggraph_out,
    unsigned int *components,
    const Csr<VertexId, Value, SizeT> &csr_graph,
    const int    max_grid_size,
    const int    num_gpus) {

    // Define CCProblem
    typedef CCProblem <
        VertexId,
        SizeT,
        Value,
        true > Problem; //use double buffer

    // Allocate host-side label array for gpu-computed results
    VertexId *h_component_ids
        = (VertexId*)malloc(sizeof(VertexId) * csr_graph.nodes);

    // Allocate CC enactor map
    CCEnactor<false> cc_enactor(false);

    // Allocate problem on GPU
    Problem *csr_problem = new Problem;
    util::GRError(csr_problem->Init(
                      false,
                      csr_graph,
                      num_gpus),
                  "CC Problem Initialization Failed", __FILE__, __LINE__);

    // Reset CC Problem Data
    util::GRError(csr_problem->Reset(
                      cc_enactor.GetFrontierType()),
                  "CC Problem Data Reset Failed", __FILE__, __LINE__);

    // Perform Connected Component
    GpuTimer gpu_timer;
    gpu_timer.Start();
    // Lunch CC Enactor
    util::GRError(cc_enactor.template Enact<Problem>(
                      csr_problem, max_grid_size),
                  "CC Problem Enact Failed", __FILE__, __LINE__);
    gpu_timer.Stop();
    float elapsed = gpu_timer.ElapsedMillis();

    // Copy out results back to Host Device
    util::GRError(csr_problem->Extract(h_component_ids),
                  "CC Problem Data Extraction Failed", __FILE__, __LINE__);

    // Compute number of components in graph
    unsigned int temp = csr_problem->num_components;
    *components = temp;

    // copy component_id per node to GunrockGraph struct
    ggraph_out->node_values = (int*)&h_component_ids[0];

    printf("GPU Connected Component finished in %lf msec.\n", elapsed);

    // Cleanup
    if (csr_problem)  delete csr_problem;

    hipDeviceSynchronize();
}

/**
 * @brief dispatch function to handle data_types
 *
 * @param[out] ggraph_out GunrockGraph type output
 * @param[in]  ggraph_in  GunrockGraph type input graph
 * @param[in]  cc_config  cc specific configurations
 * @param[in]  data_type  data type configurations
 */
void dispatch_cc(
    GunrockGraph          *ggraph_out,
    unsigned int          *components,
    const GunrockGraph    *ggraph_in,
    const GunrockConfig   cc_config,
    const GunrockDataType data_type) {
    switch (data_type.VTXID_TYPE) {
    case VTXID_INT: {
        switch (data_type.SIZET_TYPE) {
        case SIZET_INT: {
            switch (data_type.VALUE_TYPE) {
            case VALUE_INT: {
                // template type = <int, int, int>
                // build input csr format graph
                Csr<int, int, int> csr_graph(false);
                csr_graph.nodes = ggraph_in->num_nodes;
                csr_graph.edges = ggraph_in->num_edges;
                csr_graph.row_offsets    = (int*)ggraph_in->row_offsets;
                csr_graph.column_indices = (int*)ggraph_in->col_indices;

                int max_grid_size = 0; //!< 0: leave it up to the enactor
                int num_gpus      = 1; //!< number of GPUs

                // lunch cc dispatch function
                run_cc<int, int, int>(
                    ggraph_out,
                    (unsigned int*)components,
                    csr_graph,
                    max_grid_size,
                    num_gpus);

                // reset for free memory
                csr_graph.row_offsets    = NULL;
                csr_graph.column_indices = NULL;
                break;
            }
            case VALUE_UINT: {
                // template type = <int, uint, int>
                printf("Not Yet Support This DataType Combination.\n");
                break;
            }
            case VALUE_FLOAT: {
                // template type = <int, float, int>
                printf("Not Yet Support This DataType Combination.\n");
                break;
            }
            }
            break;
        }
        }
        break;
    }
    }
}

/*
 * @brief gunrock_cc function
 *
 * @param[out] ggraph_out output subgraph of cc problem
 * @param[in]  ggraph_in  input graph need to process on
 * @param[in]  cc_configs primitive specific configurations
 * @param[in]  data_type  gunrock data_type struct
 */
void gunrock_cc_func(
    GunrockGraph          *ggraph_out,
    unsigned int          *components,
    const GunrockGraph    *ggraph_in,
    const GunrockConfig   cc_configs,
    const GunrockDataType data_type) {

    // lunch dispatch function
    dispatch_cc(ggraph_out, components, ggraph_in, cc_configs, data_type);
}

// Leave this at the end of the file
// Local Variables:
// mode:c++
// c-file-style: "NVIDIA"
// End:
