// ----------------------------------------------------------------
// Gunrock -- Fast and Efficient GPU Graph Library
// ----------------------------------------------------------------
// This source code is distributed under the terms of LICENSE.TXT
// in the root directory of this source distribution.
// ----------------------------------------------------------------

/**
 * @file
 * test_bfs.cu
 *
 * @brief Simple test driver program for breadth-first search.
 */

#include <stdio.h>
#include <string>
#include <deque>
#include <vector>
#include <iostream>

// Utilities and correctness-checking
#include <gunrock/util/test_utils.cuh>

// Graph construction utils
#include <gunrock/graphio/market.cuh>

// BFS includes
#include <gunrock/app/bfs/bfs_enactor.cuh>
#include <gunrock/app/bfs/bfs_problem.cuh>
#include <gunrock/app/bfs/bfs_functor.cuh>

// DOBFS includes
#include <gunrock/app/dobfs/dobfs_enactor.cuh>
#include <gunrock/app/dobfs/dobfs_problem.cuh>
#include <gunrock/app/dobfs/dobfs_functor.cuh>

// Operator includes
#include <gunrock/oprtr/advance/kernel.cuh>
#include <gunrock/oprtr/filter/kernel.cuh>

#include <moderngpu.cuh>

using namespace gunrock;
using namespace gunrock::util;
using namespace gunrock::oprtr;
using namespace gunrock::app::bfs;
using namespace gunrock::app::dobfs;


/******************************************************************************
 * Defines, constants, globals
 ******************************************************************************/

//bool g_verbose;
//bool g_undirected;
//bool g_quick;
//bool g_stream_from_host;
//float g_alpha;
//float g_beta;

/******************************************************************************
 * Housekeeping Routines
 ******************************************************************************/
void Usage()
{
    printf (
        " test_dobfs <graph type> <graph type args> [--device=<device_index>]\n"
        " [--src=<source_index>] [--instrumented] [--idempotence=<0|1>] [--v]\n"
        " [--undirected] [--iteration-num=<num>] [--quick=<0|1>] [--mark-pred]\n"
        " [--queue-sizing=<scale factor>]\n"
        "\n"
        "Graph types and args:\n"
        "  market <file>\n"
        "    Reads a Matrix-Market coordinate-formatted graph of directed / undirected\n"
        "    edges from stdin (or from the optionally-specified file).\n"
        "  --device=<device_index>   Set GPU device for running the test. [Default: 0].\n"
        "  --undirected              Treat the graph as undirected (symmetric).\n"
        "  --idempotence=<0 or 1>    Enable: 1, Disable: 0 [Default: Enable].\n"
        "  --instrumented            Keep kernels statics [Default: Disable].\n"
        "                            total_queued, search_depth and barrier duty\n"
        "                            (a relative indicator of load imbalance.)\n"
        "  --src=<source vertex id>  Begins BFS from the source [Default: 0].\n"
        "                            If randomize: from a random source vertex.\n"
        "                            If largestdegree: from largest degree vertex.\n"
        "  --quick=<0 or 1>          Skip the CPU validation: 1, or not: 0 [Default: 1].\n"
        "  --mark-pred               Keep both label info and predecessor info.\n"
        "  --queue-sizing=<factor>   Allocates a frontier queue sized at: \n"
        "                            (graph-edges * <scale factor>). [Default: 1.0]\n"
        "  --v                       Print verbose per iteration debug info.\n"
        "  --iteration-num=<number>  Number of runs to perform the test [Default: 1].\n"
        );
}

/**
 * @brief Displays the BFS result (i.e., distance from source)
 *
 * @param[in] source_path Search depth from the source for each node.
 * @param[in] preds Predecessor node id for each node.
 * @param[in] nodes Number of nodes in the graph.
 * @param[in] MARK_PREDECESSORS Whether to show predecessor of each node.
 * @param[in] ENABLE_IDEMPOTENCE Whether to enable idempotence mode.
 */
template<typename VertexId, typename SizeT>
void DisplaySolution (VertexId *source_path,
                      VertexId *preds,
                      SizeT nodes,
                      bool MARK_PREDECESSORS,
                      bool ENABLE_IDEMPOTENCE)
{
    if (nodes > 40) nodes = 40;
    printf("\nFirst %d labels of the GPU result.\n", nodes);

    printf("[");
    for (VertexId i = 0; i < nodes; ++i) {
        PrintValue(i);
        printf(":");
        PrintValue(source_path[i]);
        if (MARK_PREDECESSORS && !ENABLE_IDEMPOTENCE) {
            printf(",");
            PrintValue(preds[i]);
        }
        printf(" ");
    }
    printf("]\n");
}

/**
 * Performance/Evaluation statistics
 */

struct Stats {
    const char *name;
    Statistic rate;
    Statistic search_depth;
    Statistic redundant_work;
    Statistic duty;

    Stats() : name(NULL), rate(), search_depth(), redundant_work(), duty() {}
    Stats(const char *name) : name(name), rate(), search_depth(), redundant_work(), duty() {}
};

struct Test_Parameter : gunrock::app::TestParameter_Base {
public:
    bool          mark_predecessors ;// Whether or not to mark src-distance vs. parent vertices
    bool          enable_idempotence;// Whether or not to enable idempotence operation
    double        max_queue_sizing1 ;
    void         *inv_graph         ;
    float         alpha;
    float         beta;

    Test_Parameter()
    {
        mark_predecessors  = false;
        enable_idempotence = false;
        max_queue_sizing1  = -1.0 ;
        inv_graph          = NULL ;
        alpha              = 0.0f;
        beta               = 0.0f;
    }

    ~Test_Parameter()
    {
    }

    void Init(CommandLineArgs &args)
    {
        TestParameter_Base::Init(args);
        mark_predecessors  = args.CheckCmdLineFlag("mark-pred");
        enable_idempotence = args.CheckCmdLineFlag("idempotence");
        args.GetCmdLineArgument("queue-sizing1", max_queue_sizing1);
        args.GetCmdLineArgument("alpha", alpha);
        args.GetCmdLineArgument("beta", beta);

        if (alpha == 0.0f)
            alpha = 6.0f;
        if (beta == 0.0f)
            beta = 6.0f;

        printf("alpha:%5f, beta:%5f\n", alpha, beta);
   }
};

/**
 * @brief Displays timing and correctness statistics
 *
 * @tparam MARK_PREDECESSORS
 * @tparam VertexId
 * @tparam Value
 * @tparam SizeT
 *
 * @param[in] stats Reference to the Stats object defined in RunTests
 * @param[in] src Source node where BFS starts
 * @param[in] h_labels Host-side vector stores computed labels for validation
 * @param[in] graph Reference to the CSR graph we process on
 * @param[in] elapsed Total elapsed kernel running time
 * @param[in] search_depth Maximum search depth of the BFS algorithm
 * @param[in] total_queued Total element queued in BFS kernel running process
 * @param[in] avg_duty Average duty of the BFS kernels
 */
template<
    bool MARK_PREDECESSORS,
    typename VertexId,
    typename Value,
    typename SizeT>
void DisplayStats(
    Stats               &stats,
    VertexId            src,
    VertexId            *h_labels,
    const Csr<VertexId, Value, SizeT> &graph,
    double              elapsed,
    VertexId            search_depth,
    long long           total_queued,
    double              avg_duty)
{
    // Compute nodes and edges visited
    SizeT edges_visited = 0;
    SizeT nodes_visited = 0;
    for (VertexId i = 0; i < graph.nodes; ++i) {
        if (h_labels[i] > -1) {
            ++nodes_visited;
            edges_visited += graph.row_offsets[i+1] - graph.row_offsets[i];
        }
    }

    double redundant_work = 0.0;
    if (total_queued > 0) {
        redundant_work = ((double) total_queued - edges_visited) / edges_visited;
        // measure duplicate edges put through queue
    }
    redundant_work *= 100;

    // Display test name
    printf("[%s] finished. ", stats.name);

    // Display statistics
    if (nodes_visited < 5) {
        printf("Fewer than 5 vertices visited.\n");
    } else {
        // Display the specific sample statistics
        double m_teps = (double) edges_visited / (elapsed * 1000.0);
        printf("\n elapsed: %.4f ms, rate: %.4f MiEdges/s", elapsed, m_teps);
        if (search_depth != 0) printf(", search_depth: %lld", (long long) search_depth);
        if (avg_duty != 0) {
            printf("\n avg CTA duty: %.2f%%", avg_duty * 100);
        }
        printf("\n src: %lld, nodes_visited: %lld, edges_visited: %lld",
               (long long) src, (long long) nodes_visited, (long long) edges_visited);
        if (total_queued > 0) {
            printf(", total queued: %lld", total_queued);
        }
        if (redundant_work > 0) {
            printf(", redundant work: %.2f%%", redundant_work);
        }
        printf("\n");
    }

}

/******************************************************************************
 * BFS Testing Routines
 *****************************************************************************/

/**
 * @brief A simple CPU-based reference BFS ranking implementation.
 *
 * @tparam VertexId
 * @tparam Value
 * @tparam SizeT
 *
 * @param[in] graph Reference to the CSR graph we process on
 * @param[in] source_path Host-side vector to store CPU computed labels for each node
 * @param[in] src Source node where BFS starts
 */
template<
    typename VertexId,
    typename Value,
    typename SizeT>
void SimpleReferenceBfs(
    const Csr<VertexId, Value, SizeT>       &graph,
    VertexId                                *source_path,
    VertexId                                src)
{
    // Initialize distances
    for (VertexId i = 0; i < graph.nodes; ++i)
    {
        source_path[i] = -1;
    }
    source_path[src] = 0;
    VertexId search_depth = 0;

    // Initialize queue for managing previously-discovered nodes
    std::deque<VertexId> frontier;
    frontier.push_back(src);

    //
    //Perform BFS
    //

    CpuTimer cpu_timer;
    cpu_timer.Start();
    while (!frontier.empty())
    {

        // Dequeue node from frontier
        VertexId dequeued_node = frontier.front();
        frontier.pop_front();
        VertexId neighbor_dist = source_path[dequeued_node] + 1;

        // Locate adjacency list
        int edges_begin = graph.row_offsets[dequeued_node];
        int edges_end = graph.row_offsets[dequeued_node + 1];

        for (int edge = edges_begin; edge < edges_end; ++edge)
        {
            //Lookup neighbor and enqueue if undiscovered
            VertexId neighbor = graph.column_indices[edge];
            if (source_path[neighbor] == -1)
            {
                source_path[neighbor] = neighbor_dist;
                if (search_depth < neighbor_dist)
                {
                    search_depth = neighbor_dist;
                }
                frontier.push_back(neighbor);
            }
        }
    }

    cpu_timer.Stop();
    float elapsed = cpu_timer.ElapsedMillis();
    search_depth++;

    printf("CPU BFS finished in %lf msec. Search depth is: %d\n",
           elapsed, search_depth);
}

/**
 * @brief Run BFS tests
 *
 * @tparam VertexId
 * @tparam Value
 * @tparam SizeT
 * @tparam INSTRUMENT
 * @tparam MARK_PREDECESSORS
 * @tparam ENABLE_IDEMPOTENCE
 *
 * @param[in] graph Reference to the CSR graph we process on
 * @param[in] inv_graph Reference to the inverse CSC graph we process on
 * @param[in] src Source node where BFS starts
 * @param[in] max_grid_size Maximum CTA occupancy
 * @param[in] num_gpus Number of GPUs
 * @param[in] max_queue_sizing Scaling factor used in edge mapping
 * @param[in] alpha Tuning parameter for switching to reverse bfs
 * @param[in] beta Tuning parameter for switching back to normal bfs
 * @param[in] iterations Number of iterations for running the test
 * @param[in] context CudaContext pointer for moderngpu APIs
 */
template <
    typename VertexId,
    typename Value,
    typename SizeT,
    bool INSTRUMENT,
    bool DEBUG,
    bool SIZE_CHECK,
    bool MARK_PREDECESSORS,
    bool ENABLE_IDEMPOTENCE>
void RunTests(Test_Parameter *parameter)
    /*Csr<VertexId, Value, SizeT> &graph,
    Csr<VertexId, Value, SizeT> &inv_graph,
    VertexId src,
    int max_grid_size,
    int num_gpus,
    double max_queue_sizing,
    float alpha,        // Tuning parameter for switching to reverse bfs
    float beta,         // Tuning parameter for switching back to normal bfs
    ContextPtr context,
    int *gpu_idx)*/
{
    typedef DOBFSProblem<
        VertexId,
        SizeT,
        Value,
        MARK_PREDECESSORS,
        ENABLE_IDEMPOTENCE,
        (MARK_PREDECESSORS && ENABLE_IDEMPOTENCE)> Problem; // does not use double buffer

    Csr<VertexId, Value, SizeT>
                 *graph                 = (Csr<VertexId, Value, SizeT>*)parameter->graph;
    Csr<VertexId, Value, SizeT>
                 *inv_graph             = (Csr<VertexId, Value, SizeT>*)parameter->inv_graph;
    VertexId      src                   = (VertexId)parameter -> src;
    int           max_grid_size         = parameter -> max_grid_size;
    int           num_gpus              = parameter -> num_gpus;
    double        max_queue_sizing      = parameter -> max_queue_sizing;
    SizeT         iterations            = parameter -> iterations;
    ContextPtr   *context               = (ContextPtr*)parameter -> context;
    int          *gpu_idx               = parameter -> gpu_idx;
    bool          g_quick               = parameter -> g_quick;
    bool          g_stream_from_host    = parameter -> g_stream_from_host;
    bool          g_undirected          = parameter -> g_undirected;
    float         alpha                 = parameter -> alpha;
    float         beta                  = parameter -> beta;
    // Allocate host-side label array (for both reference and gpu-computed results)
    VertexId     *reference_labels      = (VertexId*)malloc(sizeof(VertexId) * graph->nodes);
    VertexId     *h_labels              = (VertexId*)malloc(sizeof(VertexId) * graph->nodes);
    VertexId     *reference_check       = (g_quick) ? NULL : reference_labels;
    VertexId     *h_preds               = NULL;
    if (MARK_PREDECESSORS) {
        h_preds = (VertexId*)malloc(sizeof(VertexId) * graph->nodes);
    }

    // Allocate BFS enactor map
    DOBFSEnactor<Problem, INSTRUMENT, DEBUG, SIZE_CHECK> dobfs_enactor(gpu_idx);

    // Allocate problem on GPU
    Problem *csr_problem = new Problem;

    util::GRError(csr_problem->Init(
        g_stream_from_host,
        g_undirected,
        *graph,
        *inv_graph,
        num_gpus,
        alpha,
        beta), "Problem DOBFS Initialization Failed", __FILE__, __LINE__);

    //
    // Compute reference CPU BFS solution for source-distance
    //
    if (reference_check != NULL)
    {
        printf("compute ref value\n");
        SimpleReferenceBfs(
                *graph,
                reference_check,
                src);
        printf("\n");
    }

    Stats *stats = new Stats("GPU DOBFS");

    long long           total_queued = 0;
    VertexId            search_depth = 0;
    double              avg_duty     = 0.0;
    float               elapsed      = 0.0f;

    // Perform BFS
    GpuTimer gpu_timer;

    for (int iter=0; iter < iterations; ++iter)
    {
        util::GRError(csr_problem->Reset(src, dobfs_enactor.GetFrontierType(), max_queue_sizing), "DOBFS Problem Data Reset Failed", __FILE__, __LINE__);
        gpu_timer.Start();
        util::GRError(dobfs_enactor.template Enact<Problem>(*context, csr_problem, src, max_grid_size), "DOBFS Problem Enact Failed", __FILE__, __LINE__);
        gpu_timer.Stop();
        elapsed += gpu_timer.ElapsedMillis();
    }
    elapsed /= iterations;

    dobfs_enactor.GetStatistics(total_queued, search_depth, avg_duty);

    // Copy out results
    util::GRError(csr_problem->Extract(h_labels, h_preds), "DOBFS Problem Data Extraction Failed", __FILE__, __LINE__);

    // Verify the result
    if (reference_check != NULL) {
        if (!MARK_PREDECESSORS) {
            printf("Validity: ");
            CompareResults(h_labels, reference_check, graph->nodes, true);
        }
    }
    printf("\nFirst 40 labels of the GPU result."); 
    // Display Solution
    DisplaySolution(h_labels, h_preds, graph->nodes, MARK_PREDECESSORS, ENABLE_IDEMPOTENCE);

    DisplayStats<MARK_PREDECESSORS>(
        *stats,
        src,
        h_labels,
        *graph,
        elapsed,
        search_depth,
        total_queued,
        avg_duty);

    // Cleanup
    delete stats;
    if (csr_problem) delete csr_problem;
    if (reference_labels) free(reference_labels);
    if (h_labels) free(h_labels);
    if (h_preds) free(h_preds);

    hipDeviceSynchronize();
}

template <
    typename    VertexId,
    typename    Value,
    typename    SizeT,
    bool        INSTRUMENT,
    bool        DEBUG,
    bool        SIZE_CHECK,
    bool        MARK_PREDECESSORS>
void RunTests_enable_idempotence(Test_Parameter *parameter)
{
    if (parameter->enable_idempotence) RunTests
        <VertexId, Value, SizeT, INSTRUMENT, DEBUG, SIZE_CHECK, MARK_PREDECESSORS,
        true > (parameter);
   else RunTests
        <VertexId, Value, SizeT, INSTRUMENT, DEBUG, SIZE_CHECK, MARK_PREDECESSORS,
        false> (parameter);
}

template <
    typename    VertexId,
    typename    Value,
    typename    SizeT,
    bool        INSTRUMENT,
    bool        DEBUG,
    bool        SIZE_CHECK>
void RunTests_mark_predecessors(Test_Parameter *parameter)
{
    if (parameter->mark_predecessors) RunTests_enable_idempotence
        <VertexId, Value, SizeT, INSTRUMENT, DEBUG, SIZE_CHECK,
        true > (parameter);
   else RunTests_enable_idempotence
        <VertexId, Value, SizeT, INSTRUMENT, DEBUG, SIZE_CHECK,
        false> (parameter);
}

template <
    typename      VertexId,
    typename      Value,
    typename      SizeT,
    bool          INSTRUMENT,
    bool          DEBUG>
void RunTests_size_check(Test_Parameter *parameter)
{
    if (parameter->size_check) RunTests_mark_predecessors
        <VertexId, Value, SizeT, INSTRUMENT, DEBUG,
        true > (parameter);
   else RunTests_mark_predecessors
        <VertexId, Value, SizeT, INSTRUMENT, DEBUG,
        false> (parameter);
}

template <
    typename    VertexId,
    typename    Value,
    typename    SizeT,
    bool        INSTRUMENT>
void RunTests_debug(Test_Parameter *parameter)
{
    if (parameter->debug) RunTests_size_check
        <VertexId, Value, SizeT, INSTRUMENT,
        true > (parameter);
    else RunTests_size_check
        <VertexId, Value, SizeT, INSTRUMENT,
        false> (parameter);
}

template <
    typename      VertexId,
    typename      Value,
    typename      SizeT>
void RunTests_instrumented(Test_Parameter *parameter)
{
    if (parameter->instrumented) RunTests_debug
        <VertexId, Value, SizeT,
        true > (parameter);
    else RunTests_debug
        <VertexId, Value, SizeT,
        false> (parameter);
}

template <
    typename VertexId,
    typename Value,
    typename SizeT>
void RunTests(
    Csr<VertexId, Value, SizeT> *graph,
    Csr<VertexId, Value, SizeT> *inv_graph,
    CommandLineArgs             &args,
    int                          num_gpus,
    ContextPtr                  *context,
    int                         *gpu_idx,
    hipStream_t                *streams = NULL)
{
    string src_str="";
    Test_Parameter *parameter = new Test_Parameter;   
 
    parameter -> Init(args);
    parameter -> graph              = graph;
    parameter -> inv_graph          = inv_graph;
    parameter -> num_gpus           = num_gpus;
    parameter -> context            = context;
    parameter -> gpu_idx            = gpu_idx;
    parameter -> streams            = streams;

    args.GetCmdLineArgument("src", src_str);
    if (src_str.empty()) {
        parameter->src = 0;
    } else if (src_str.compare("randomize") == 0) {
        parameter->src = graphio::RandomNode(graph->nodes);
    } else if (src_str.compare("largestdegree") == 0) {
        int temp;
        parameter->src = graph->GetNodeWithHighestDegree(temp);
    } else {
        args.GetCmdLineArgument("src", parameter->src);
    }   
    printf("src = %lld\n", (long long) parameter->src);

    RunTests_instrumented<VertexId, Value, SizeT>(parameter);
}

/******************************************************************************
* Main
******************************************************************************/

int main( int argc, char** argv)
{
    CommandLineArgs args(argc, argv);

    if ((argc < 2) || (args.CheckCmdLineFlag("help")))
    {
        Usage();
        return 1;
    }

    //DeviceInit(args);
    //hipSetDeviceFlags(hipDeviceMapHost);

    int dev = 0;
    args.GetCmdLineArgument("device", dev);
    ContextPtr context = mgpu::CreateCudaDevice(dev);

    //srand(0); // Presently deterministic
    //srand(time(NULL));

	// Parse graph-contruction params
	bool g_undirected = args.CheckCmdLineFlag("undirected");

    std::string graph_type = argv[1];
    int flags = args.ParsedArgc();
    int graph_args = argc - flags - 1;

    if (graph_args < 1)
    {
        Usage();
        return 1;
    }

    //
    // Construct graph and perform search(es)
    //

    if (graph_type == "market")
    {
        // Matrix-market coordinate-formatted graph file

        typedef int VertexId;                   // Use as the node identifier
        typedef int Value;                      // Use as the value type
        typedef int SizeT;                      // Use as the graph size type
        Csr<VertexId, Value, SizeT> csr(false); // default for stream_from_host

        Csr<VertexId, Value, SizeT> inv_csr(false);

        if (graph_args < 1) { Usage(); return 1; }

        char *market_filename = (graph_args == 2) ? argv[2] : NULL;
        if (graphio::BuildMarketGraph<false>(
                market_filename,
                csr,
                g_undirected,
                false) != 0)
        {
            return 1;
        }

        if (!g_undirected)
        {
            if (graphio::BuildMarketGraph<false>(
                    market_filename,
                    inv_csr,
                    g_undirected,
                    true) != 0)
            {
                return 1;
            }
        }

        csr.PrintHistogram();

        RunTests<VertexId, Value, SizeT>(&csr, g_undirected? &csr : &inv_csr, args, 1, &context, &dev);
    }
    else
    {
        fprintf(stderr, "Unspecified graph type\n");
        return 1;
    }
    return 0;
}
