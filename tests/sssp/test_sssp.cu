// ----------------------------------------------------------------
// Gunrock -- Fast and Efficient GPU Graph Library
// ----------------------------------------------------------------
// This source code is distributed under the terms of LICENSE.TXT
// in the root directory of this source distribution.
// ----------------------------------------------------------------

/**
 * @file
 * test_bfs.cu
 *
 * @brief Simple test driver program for breadth-first search.
 */

#include <stdio.h> 
#include <string>
#include <deque>
#include <vector>
#include <iostream>

// Utilities and correctness-checking
#include <gunrock/util/test_utils.cuh>

// Graph construction utils
#include <gunrock/graphio/market.cuh>

// SSSP includes
#include <gunrock/app/sssp/sssp_enactor.cuh>
#include <gunrock/app/sssp/sssp_problem.cuh>
#include <gunrock/app/sssp/sssp_functor.cuh>

// Operator includes
#include <gunrock/oprtr/advance/kernel.cuh>
#include <gunrock/oprtr/filter/kernel.cuh>
#include <gunrock/priority_queue/kernel.cuh>

#include <moderngpu.cuh>

// Boost includes for CPU dijkstra SSSP reference algorithms
#include <boost/config.hpp>
#include <boost/graph/graph_traits.hpp>
#include <boost/graph/adjacency_list.hpp>
#include <boost/graph/dijkstra_shortest_paths.hpp>
#include <boost/property_map/property_map.hpp>

using namespace gunrock;
using namespace gunrock::util;
using namespace gunrock::oprtr;
using namespace gunrock::app::sssp;


/******************************************************************************
 * Defines, constants, globals 
 ******************************************************************************/

bool g_verbose;
bool g_undirected;
bool g_quick;
bool g_stream_from_host;

/******************************************************************************
 * Housekeeping Routines
 ******************************************************************************/
 void Usage()
 {
 printf("\ntest_sssp <graph type> <graph type args> [--device=<device_index>] "
        "[--undirected] [--instrumented] [--src=<source index>] [--quick]\n"
        "[--v] [mark-pred] [--queue-sizing=<scale factor>]\n"
        "\n"
        "Graph types and args:\n"
        "  market [<file>]\n"
        "    Reads a Matrix-Market coordinate-formatted graph of directed/undirected\n"
        "    edges from stdin (or from the optionally-specified file).\n"
        "  --device=<device_index>  Set GPU device for running the graph primitive.\n"
        "  --undirected If set then treat the graph as undirected.\n"
        "  --instrumented If set then kernels keep track of queue-search_depth\n"
        "  and barrier duty (a relative indicator of load imbalance.)\n"
        "  --src Begins SSSP from the vertex <source index>. If set as randomize\n"
        "  then will begin with a random source vertex.\n"
        "  If set as largestdegree then will begin with the node which has\n"
        "  largest degree.\n"
        "  --quick If set will skip the CPU validation code.\n"
        "  --v Whether to show debug info.\n"
        "  --mark-pred If set then keep not only label info but also predecessor info.\n"
        "  --queue-sizing Allocates a frontier queue sized at (graph-edges * <scale factor>).\n"
        );
 }

 /**
  * @brief Displays the SSSP result (i.e., distance from source)
  *
  * @param[in] source_path Search depth from the source for each node.
  * @param[in] preds Predecessor node id for each node.
  * @param[in] nodes Number of nodes in the graph.
  * @param[in] MARK_PREDECESSORS Whether to show predecessor of each node.
  */
 template<typename VertexId, typename SizeT>
 void DisplaySolution(VertexId *source_path, SizeT nodes)
 {
    if (nodes > 40)
        nodes = 40;
    printf("[");
    for (VertexId i = 0; i < nodes; ++i) {
        PrintValue(i);
        printf(":");
        PrintValue(source_path[i]);
        printf(" ");
    }
    printf("]\n");
 }

 /**
  * Performance/Evaluation statistics
  */ 

struct Stats {
    char *name;
    Statistic rate;
    Statistic search_depth;
    Statistic redundant_work;
    Statistic duty;

    Stats() : name(NULL), rate(), search_depth(), redundant_work(), duty() {}
    Stats(char *name) : name(name), rate(), search_depth(), redundant_work(), duty() {}
};

/**
 * @brief Displays timing and correctness statistics
 *
 * @tparam MARK_PREDECESSORS
 * @tparam VertexId
 * @tparam Value
 * @tparam SizeT
 * 
 * @param[in] stats Reference to the Stats object defined in RunTests
 * @param[in] src Source node where SSSP starts
 * @param[in] h_labels Host-side vector stores computed labels for validation
 * @param[in] graph Reference to the CSR graph we process on
 * @param[in] elapsed Total elapsed kernel running time
 * @param[in] search_depth Maximum search depth of the SSSP algorithm
 * @param[in] total_queued Total element queued in SSSP kernel running process
 * @param[in] avg_duty Average duty of the SSSP kernels
 */
template<
    typename VertexId,
    typename Value,
    typename SizeT>
void DisplayStats(
    Stats               &stats,
    VertexId            src,
    Value               *h_labels,
    const Csr<VertexId, Value, SizeT> &graph,
    double              elapsed,
    VertexId            search_depth,
    long long           total_queued,
    double              avg_duty)
{
    // Compute nodes and edges visited
    SizeT edges_visited = 0;
    SizeT nodes_visited = 0;
    for (VertexId i = 0; i < graph.nodes; ++i) {
        if (h_labels[i] < UINT_MAX) {
            ++nodes_visited;
            edges_visited += graph.row_offsets[i+1] - graph.row_offsets[i];
        }
    }

    double redundant_work = 0.0;
    if (total_queued > 0) {
        redundant_work = ((double) total_queued - edges_visited) / edges_visited;        // measure duplicate edges put through queue
    }
    redundant_work *= 100;

    // Display test name
    printf("[%s] finished. ", stats.name);

    // Display statistics
    if (nodes_visited < 5) {
        printf("Fewer than 5 vertices visited.\n");
    } else {
        // Display the specific sample statistics
        double m_teps = (double) edges_visited / (elapsed * 1000.0);
        printf(" elapsed: %.3f ms, rate: %.3f MiEdges/s", elapsed, m_teps);
        if (search_depth != 0) printf(", search_depth: %lld", (long long) search_depth);
        if (avg_duty != 0) {
            printf("\n avg CTA duty: %.2f%%", avg_duty * 100);
        }
        printf("\n src: %lld, nodes_visited: %lld, edges visited: %lld",
            (long long) src, (long long) nodes_visited, (long long) edges_visited);
        if (total_queued > 0) {
            printf(", total queued: %lld", total_queued);
        }
        if (redundant_work > 0) {
            printf(", redundant work: %.2f%%", redundant_work);
        }
        printf("\n");
    }
}

/******************************************************************************
 * SSSP Testing Routines
 *****************************************************************************/

 /**
  * @brief A simple CPU-based reference SSSP ranking implementation.
  *
  * @tparam VertexId
  * @tparam Value
  * @tparam SizeT
  *
  * @param[in] graph Reference to the CSR graph we process on
  * @param[in] source_path Host-side vector to store CPU computed labels for each node
  * @param[in] src Source node where SSSP starts
  */
 template<
    typename VertexId,
    typename Value,
    typename SizeT,
    bool     MARK_PREDECESSORS>
void SimpleReferenceSssp(
    const Csr<VertexId, Value, SizeT>       &graph,
    Value    *node_values,
    VertexId *node_preds,
    VertexId src)
{
    using namespace boost; 
    // Prepare Boost Datatype and Data structure
    typedef adjacency_list<vecS, vecS, directedS,
            no_property, property <edge_weight_t, int> > Graph;
    typedef graph_traits<Graph>::vertex_descriptor vertex_descriptor;
    typedef graph_traits<Graph>::edge_descriptor edge_descriptor;

    typedef std::pair<VertexId, VertexId> Edge;

    Edge* edges = (Edge*)malloc(sizeof(Edge)*graph.edges);
    Value *weight = (Value*)malloc(sizeof(Value)*graph.edges);

    for (int i = 0; i < graph.nodes; ++i)
    {
        for (int j = graph.row_offsets[i]; j < graph.row_offsets[i+1]; ++j)
        {
            edges[j] = Edge(i, graph.column_indices[j]);
            weight[j] = graph.edge_values[j];
        }
    }

    Graph g(edges, edges + graph.edges, weight, graph.nodes);

    std::vector<Value> d(graph.nodes);
    std::vector<vertex_descriptor> p(graph.nodes);
    vertex_descriptor s = vertex(src, g);

    property_map<Graph, vertex_index_t>::type indexmap = get(vertex_index, g);

    //
    // Perform SSSP
    //

    CpuTimer cpu_timer;
    cpu_timer.Start();

    if (MARK_PREDECESSORS)
        dijkstra_shortest_paths(g,
                            s,
                            predecessor_map(boost::make_iterator_property_map(p.begin(), get(boost::vertex_index, g))).
                            distance_map(boost::make_iterator_property_map(d.begin(), get(boost::vertex_index, g))));
    else
        dijkstra_shortest_paths(g,
                            s,
                            distance_map(boost::make_iterator_property_map(d.begin(), get(boost::vertex_index, g))));
    cpu_timer.Stop();
    float elapsed = cpu_timer.ElapsedMillis();

    printf("CPU SSSP finished in %lf msec.\n", elapsed);

    Coo<unsigned int, unsigned int>* sort_dist = NULL;
    Coo<unsigned int, unsigned int>* sort_pred = NULL;
    sort_dist = (Coo<unsigned int, unsigned int>*)malloc(sizeof(Coo<unsigned int, unsigned int>) * graph.nodes);
    if (MARK_PREDECESSORS)
        sort_pred = (Coo<unsigned int, unsigned int>*)malloc(sizeof(Coo<unsigned int, unsigned int>) * graph.nodes);

    graph_traits < Graph >::vertex_iterator vi, vend;
    for (tie(vi, vend) = vertices(g); vi != vend; ++vi)
    {
        sort_dist[(*vi)].row = (*vi);
        sort_dist[(*vi)].col = d[(*vi)];  
    }
    std::stable_sort(sort_dist, sort_dist + graph.nodes, RowFirstTupleCompare<Coo<unsigned int, unsigned int> >);

    if (MARK_PREDECESSORS) {
        for (tie(vi, vend) = vertices(g); vi != vend; ++vi)
        {
            sort_pred[(*vi)].row = (*vi);
            sort_pred[(*vi)].col = p[(*vi)];
        }
        std::stable_sort(sort_pred, sort_pred + graph.nodes, RowFirstTupleCompare<Coo<unsigned int, unsigned int> >);
    }

    for (int i = 0; i < graph.nodes; ++i) {
        node_values[i] = sort_dist[i].col;
    }
    if (MARK_PREDECESSORS)
        for (int i = 0; i < graph.nodes; ++i) {
            node_preds[i] = sort_pred[i].col;
        }

    free(sort_dist);
    if (MARK_PREDECESSORS) free(sort_pred);
}

/**
 * @brief Run SSSP tests
 *
 * @tparam VertexId
 * @tparam Value
 * @tparam SizeT
 * @tparam INSTRUMENT
 * @tparam MARK_PREDECESSORS
 *
 * @param[in] graph Reference to the CSR graph we process on
 * @param[in] src Source node where SSSP starts
 * @param[in] max_grid_size Maximum CTA occupancy
 * @param[in] num_gpus Number of GPUs
 * @param[in] max_queue_sizing Scaling factor used in edge mapping
 *
 */
template <
    typename VertexId,
    typename Value,
    typename SizeT,
    bool INSTRUMENT,
    bool MARK_PREDECESSORS>
void RunTests(
    Csr<VertexId, Value, SizeT> &graph,
    VertexId    src,
    int         max_grid_size,
    int         num_gpus,
    float       queue_sizing,
    ContextPtr  *context,
    std::string partition_method,
    int         *gpu_idx)
{
        typedef SSSPProblem<
            VertexId,
            SizeT,
            Value,
            MARK_PREDECESSORS> Problem;

        // Allocate host-side label array (for both reference and gpu-computed results)
        Value     *reference_labels       = new Value[graph.nodes];
        Value     *h_labels               = new Value[graph.nodes];
        Value     *reference_check_label  = (g_quick) ? NULL : reference_labels;
        VertexId  *reference_preds        = NULL;
        VertexId  *h_preds                = NULL;
        VertexId  *reference_check_pred   = NULL;

        if (MARK_PREDECESSORS) {
            reference_preds       = new VertexId[graph.nodes];
            h_preds               = new VertexId[graph.nodes];
            reference_check_pred  = (g_quick) ? NULL : reference_preds;
        }
            
        // Allocate SSSP enactor map
        SSSPEnactor<Problem, INSTRUMENT>* sssp_enactor
            = new SSSPEnactor<Problem, INSTRUMENT>(g_verbose, num_gpus, gpu_idx);

        // Allocate problem on GPU
        Problem *csr_problem = new Problem;
        util::GRError(csr_problem->Init(
            g_stream_from_host,
            graph,
            NULL,
            num_gpus,
            gpu_idx,
            partition_method,
            queue_sizing), "Problem SSSP Initialization Failed", __FILE__, __LINE__);

        //
        // Compute reference CPU SSSP solution for source-distance
        //
        if (reference_check_label != NULL)
        {
            printf("compute ref value\n");
            SimpleReferenceSssp<VertexId, Value, SizeT, MARK_PREDECESSORS>(
                    graph,
                    reference_check_label,
                    reference_check_pred,
                    src);
            printf("\n");
        }

        Stats      *stats       = new Stats("GPU SSSP");
        long long  total_queued = 0;
        VertexId   search_depth = 0;
        double     avg_duty     = 0.0;

        // Perform SSSP
        CpuTimer cpu_timer;

        util::GRError(csr_problem->Reset(src, sssp_enactor->GetFrontierType(), queue_sizing), "SSSP Problem Data Reset Failed", __FILE__, __LINE__); 
        cpu_timer.Start();
        util::GRError(sssp_enactor->Enact(context, csr_problem, src, queue_sizing, max_grid_size), "SSSP Problem Enact Failed", __FILE__, __LINE__);
        cpu_timer.Stop();

        sssp_enactor->GetStatistics(total_queued, search_depth, avg_duty);

        float elapsed = cpu_timer.ElapsedMillis();

        // Copy out results
        util::GRError(csr_problem->Extract(h_labels, h_preds), "SSSP Problem Data Extraction Failed", __FILE__, __LINE__);

        // Verify the result
        if (reference_check_label != NULL) {
            printf("Label Validity: ");
            CompareResults(h_labels, reference_check_label, graph.nodes, true);
        }
        
        // Display Solution
        printf("\nFirst 40 labels of the GPU result.\n"); 
        DisplaySolution(h_labels, graph.nodes);
        printf("\nFirst 40 labels of the reference CPU result.\n"); 
        DisplaySolution(reference_check_label, graph.nodes);

        if (MARK_PREDECESSORS) {
            printf("\nFirst 40 preds of the GPU result.\n"); 
            DisplaySolution(h_preds, graph.nodes);
            printf("\nFirst 40 preds of the reference CPU result (could be different because the paths are not unique).\n"); 
            DisplaySolution(reference_check_pred, graph.nodes);
        }

        DisplayStats(
            *stats,
            src,
            h_labels,
            graph,
            elapsed,
            search_depth,
            total_queued,
            avg_duty);


        // Cleanup
        if (stats           ) {delete stats           ; stats            = NULL;}
        if (sssp_enactor    ) {delete sssp_enactor    ; sssp_enactor     = NULL;}
        if (csr_problem     ) {delete csr_problem     ; csr_problem      = NULL;}
        if (reference_labels) {delete reference_labels; reference_labels = NULL;}
        if (h_labels        ) {delete h_labels        ; h_labels         = NULL;}
        if (reference_preds ) {delete reference_preds ; reference_preds  = NULL;}
        if (h_preds         ) {delete h_preds         ; h_preds          = NULL;}

        //hipDeviceSynchronize();
}

/**
 * @brief RunTests entry
 *
 * @tparam VertexId
 * @tparam Value
 * @tparam SizeT
 *
 * @param[in] graph Reference to the CSR graph we process on
 * @param[in] args Reference to the command line arguments
 */
template <
    typename VertexId,
    typename Value,
    typename SizeT>
void RunTests(
    Csr<VertexId, Value, SizeT> &graph,
    CommandLineArgs             &args,
    int                         num_gpus,
    ContextPtr                  *context,
    int                         *gpu_idx)
{
    VertexId            src                 = -1;           // Use whatever the specified graph-type's default is
    std::string         src_str;
    bool                instrumented        = false;        // Whether or not to collect instrumentation from kernels
    int                 max_grid_size       = 0;            // maximum grid size (0: leave it up to the enactor)
    //int                 num_gpus            = 1;            // Number of GPUs for multi-gpu enactor to use
    float               max_queue_sizing    = 1.0;
    bool                mark_pred           = false;
    std::string         partition_method    = "random";

    instrumented = args.CheckCmdLineFlag("instrumented");
    args.GetCmdLineArgument("src", src_str);
    if (src_str.empty()) {
        src = 0;
    } else if (src_str.compare("randomize") == 0) {
        src = graphio::RandomNode(graph.nodes);
    } else if (src_str.compare("largestdegree") == 0) {
        src = graph.GetNodeWithHighestDegree();
    } else {
        args.GetCmdLineArgument("src", src);
    }

    mark_pred = args.CheckCmdLineFlag("mark-pred");
    args.GetCmdLineArgument("queue-sizing", max_queue_sizing);

    //printf("Display neighbor list of src:\n");
    //graph.DisplayNeighborList(src);

    g_quick = args.CheckCmdLineFlag("quick");
    g_verbose = args.CheckCmdLineFlag("v");
    if (args.CheckCmdLineFlag  ("partition_method")) 
        args.GetCmdLineArgument("partition_method",partition_method);


    if (mark_pred) {
        if (instrumented) {
            RunTests<VertexId, Value, SizeT, true, true>(
                    graph,
                    src,
                    max_grid_size,
                    num_gpus,
                    max_queue_sizing,
                    context,
                    partition_method,
                    gpu_idx);
        } else {
            RunTests<VertexId, Value, SizeT, false, true>(
                    graph,
                    src,
                    max_grid_size,
                    num_gpus,
                    max_queue_sizing,
                    context,
                    partition_method,
                    gpu_idx);
        }
    } else {
        if (instrumented) {
            RunTests<VertexId, Value, SizeT, true, false>(
                    graph,
                    src,
                    max_grid_size,
                    num_gpus,
                    max_queue_sizing,
                    context,
                    partition_method,
                    gpu_idx);
        } else {
            RunTests<VertexId, Value, SizeT, false, false>(
                    graph,
                    src,
                    max_grid_size,
                    num_gpus,
                    max_queue_sizing,
                    context,
                    partition_method,
                    gpu_idx);
        }
    }

}



/******************************************************************************
* Main
******************************************************************************/

int main( int argc, char** argv)
{
    CommandLineArgs args(argc, argv);
    int        num_gpus = 0;
    int        *gpu_idx = NULL;
    ContextPtr *context = NULL;

    if ((argc < 2) || (args.CheckCmdLineFlag("help"))) {
        Usage();
        return 1;
    }

    //DeviceInit(args);
    //hipSetDeviceFlags(hipDeviceMapHost);
    if (args.CheckCmdLineFlag  ("device"))
    {
        std::vector<int> gpus;
        args.GetCmdLineArguments<int>("device",gpus);
        num_gpus   = gpus.size();
        gpu_idx    = new int[num_gpus];
        for (int i=0;i<num_gpus;i++)
            gpu_idx[i] = gpus[i];
    } else {
        num_gpus   = 1;
        gpu_idx    = new int[num_gpus];
        gpu_idx[0] = 0;
    }
    context  = new ContextPtr[num_gpus];
    printf("Using %d gpus: ", num_gpus);
    for (int i=0;i<num_gpus;i++)
    {
        printf(" %d ", gpu_idx[i]);
        context[i] = mgpu::CreateCudaDevice(gpu_idx[i]);
    }
    printf("\n"); fflush(stdout);
    
    //srand(0);									// Presently deterministic
    //srand(time(NULL));

    // Parse graph-contruction params
    g_undirected = args.CheckCmdLineFlag("undirected");
    std::string graph_type = argv[1];
    int flags = args.ParsedArgc();
    int graph_args = argc - flags - 1;

    if (graph_args < 1) {
        Usage();
        return 1;
    }
	
    //
    // Construct graph and perform search(es)
    //

    if (graph_type == "market") {
    // Matrix-market coordinate-formatted graph file

        typedef int VertexId;                   // Use as the node identifier type
        typedef int Value;             // Use as the value type
        typedef int SizeT;                      // Use as the graph size type
        Csr<VertexId, Value, SizeT> csr(false); // default value for stream_from_host is false

        if (graph_args < 1) { Usage(); return 1; }
        char *market_filename = (graph_args == 2) ? argv[2] : NULL;
        if (graphio::BuildMarketGraph<true>(
            market_filename, 
            csr, 
            g_undirected,
            false) != 0) // no inverse graph
        {
            return 1;
        }

        csr.PrintHistogram();
        csr.DisplayGraph(true); //print graph with edge_value
        
        csr.GetAverageEdgeValue();
        csr.GetAverageDegree();
		
        // Run tests
        RunTests(csr, args, num_gpus, context, gpu_idx);

    } else {
        // Unknown graph type
        fprintf(stderr, "Unspecified graph type\n");
        return 1;
    }

    return 0;
}
