#include "hip/hip_runtime.h"
// ----------------------------------------------------------------
// Gunrock -- Fast and Efficient GPU Graph Library
// ----------------------------------------------------------------
// This source code is distributed under the terms of LICENSE.TXT
// in the root directory of this source distribution.
// ----------------------------------------------------------------

/**
 * @file
 * test_sssp.cu
 *
 * @brief Simple test driver program for single source shorest path.
 */

#include <stdio.h> 
#include <string>
#include <deque>
#include <vector>
#include <iostream>

// Utilities and correctness-checking
#include <gunrock/util/test_utils.cuh>

// Graph construction utils
#include <gunrock/graphio/market.cuh>

// SSSP includes
#include <gunrock/app/sssp/sssp_enactor.cuh>
#include <gunrock/app/sssp/sssp_problem.cuh>
#include <gunrock/app/sssp/sssp_functor.cuh>

// Operator includes
#include <gunrock/oprtr/advance/kernel.cuh>
#include <gunrock/oprtr/filter/kernel.cuh>
#include <gunrock/priority_queue/kernel.cuh>

#include <moderngpu.cuh>

// Boost includes for CPU dijkstra SSSP reference algorithms
#include <boost/config.hpp>
#include <boost/graph/graph_traits.hpp>
#include <boost/graph/adjacency_list.hpp>
#include <boost/graph/dijkstra_shortest_paths.hpp>
#include <boost/property_map/property_map.hpp>

using namespace gunrock;
using namespace gunrock::util;
using namespace gunrock::oprtr;
using namespace gunrock::app::sssp;


/******************************************************************************
 * Defines, constants, globals 
 ******************************************************************************/

bool g_verbose;
bool g_undirected;
bool g_quick;
bool g_stream_from_host;

/******************************************************************************
 * Housekeeping Routines
 ******************************************************************************/
 void Usage()
 {
 printf("\ntest_sssp <graph type> <graph type args> [--device=<device_index>] "
        "[--undirected] [--instrumented] [--src=<source index>] [--quick]\n"
        "[--v] [mark-pred] [--queue-sizing=<scale factor>] "
        "[--in-sizing=<in/out queue scale factor>] [--disable-size-check] "
        "[--grid-size=<grid size>] [partition_method=random / biasrandom / clustered / metis]\n"
        "\n"
        "Graph types and args:\n"
        "  market [<file>]\n"
        "    Reads a Matrix-Market coordinate-formatted graph of directed/undirected\n"
        "    edges from stdin (or from the optionally-specified file).\n"
        "  --device=<device_index>  Set GPU device for running the graph primitive.\n"
        "  --undirected If set then treat the graph as undirected.\n"
        "  --instrumented If set then kernels keep track of queue-search_depth\n"
        "  and barrier duty (a relative indicator of load imbalance.)\n"
        "  --src Begins SSSP from the vertex <source index>. If set as randomize\n"
        "  then will begin with a random source vertex.\n"
        "  If set as largestdegree then will begin with the node which has\n"
        "  largest degree.\n"
        "  --quick If set will skip the CPU validation code.\n"
        "  --v Whether to show debug info.\n"
        "  --mark-pred If set then keep not only label info but also predecessor info.\n"
        "  --queue-sizing Allocates a frontier queue sized at (graph-edges * <scale factor>).\n"
        );
 }

 /**
  * @brief Displays the SSSP result (i.e., distance from source)
  *
  * @param[in] source_path Search depth from the source for each node.
  * @param[in] preds Predecessor node id for each node.
  * @param[in] nodes Number of nodes in the graph.
  * @param[in] MARK_PREDECESSORS Whether to show predecessor of each node.
  */
 template<typename VertexId, typename SizeT>
 void DisplaySolution(VertexId *source_path, SizeT nodes)
 {
    if (nodes > 40)
        nodes = 40;
    printf("[");
    for (VertexId i = 0; i < nodes; ++i) {
        PrintValue(i);
        printf(":");
        PrintValue(source_path[i]);
        printf(" ");
    }
    printf("]\n");
 }

 /**
  * Performance/Evaluation statistics
  */ 

struct Stats {
    const char *name;
    Statistic rate;
    Statistic search_depth;
    Statistic redundant_work;
    Statistic duty;

    Stats() : name(NULL), rate(), search_depth(), redundant_work(), duty() {}
    Stats(const char *name) : name(name), rate(), search_depth(), redundant_work(), duty() {}
};

/**
 * @brief Displays timing and correctness statistics
 *
 * @tparam MARK_PREDECESSORS
 * @tparam VertexId
 * @tparam Value
 * @tparam SizeT
 * 
 * @param[in] stats Reference to the Stats object defined in RunTests
 * @param[in] src Source node where SSSP starts
 * @param[in] h_labels Host-side vector stores computed labels for validation
 * @param[in] graph Reference to the CSR graph we process on
 * @param[in] elapsed Total elapsed kernel running time
 * @param[in] search_depth Maximum search depth of the SSSP algorithm
 * @param[in] total_queued Total element queued in SSSP kernel running process
 * @param[in] avg_duty Average duty of the SSSP kernels
 */
template<
    typename VertexId,
    typename Value,
    typename SizeT>
void DisplayStats(
    Stats               &stats,
    VertexId            src,
    Value               *h_labels,
    const Csr<VertexId, Value, SizeT> &graph,
    double              elapsed,
    VertexId            search_depth,
    long long           total_queued,
    double              avg_duty)
{
    // Compute nodes and edges visited
    SizeT edges_visited = 0;
    SizeT nodes_visited = 0;
    for (VertexId i = 0; i < graph.nodes; ++i) {
        if (h_labels[i] < util::MaxValue<VertexId>()) {
            ++nodes_visited;
            edges_visited += graph.row_offsets[i+1] - graph.row_offsets[i];
        }
    }

    double redundant_work = 0.0;
    if (total_queued > 0) {
        redundant_work = ((double) total_queued - edges_visited) / edges_visited;        // measure duplicate edges put through queue
    }
    redundant_work *= 100;

    // Display test name
    printf("[%s] finished. ", stats.name);

    // Display statistics
    if (nodes_visited < 5) {
        printf("Fewer than 5 vertices visited.\n");
    } else {
        // Display the specific sample statistics
        double m_teps = (double) edges_visited / (elapsed * 1000.0);
        printf(" elapsed: %.3f ms, rate: %.3f MiEdges/s", elapsed, m_teps);
        if (search_depth != 0) printf(", search_depth: %lld", (long long) search_depth);
        if (avg_duty != 0) {
            printf("\n avg CTA duty: %.2f%%", avg_duty * 100);
        }
        printf("\n src: %lld, nodes_visited: %lld, edges visited: %lld",
            (long long) src, (long long) nodes_visited, (long long) edges_visited);
        if (total_queued > 0) {
            printf(", total queued: %lld", total_queued);
        }
        if (redundant_work > 0) {
            printf(", redundant work: %.2f%%", redundant_work);
        }
        printf("\n");
    }
}

/******************************************************************************
 * SSSP Testing Routines
 *****************************************************************************/

 /**
  * @brief A simple CPU-based reference SSSP ranking implementation.
  *
  * @tparam VertexId
  * @tparam Value
  * @tparam SizeT
  * @tparam MARK_PREDECESSORS
  *
  * @param[in] graph Reference to the CSR graph we process on
  * @param[in] node_values Host-side vector to store CPU computed labels for each node
  * @param[in] node_preds Host-side vector to store CPU computed predecessors for each node
  * @param[in] src Source node where SSSP starts
  */
template<
    typename VertexId,
    typename Value,
    typename SizeT,
    bool     MARK_PREDECESSORS>
void SimpleReferenceSssp(
    const Csr<VertexId, Value, SizeT>       &graph,
    Value                                   *node_values,
    VertexId                                *node_preds,
    VertexId                                src)
{
    using namespace boost; 
    // Prepare Boost Datatype and Data structure
    typedef adjacency_list<vecS, vecS, directedS,
            no_property, property <edge_weight_t, unsigned int> > Graph;
    typedef graph_traits<Graph>::vertex_descriptor vertex_descriptor;
    typedef graph_traits<Graph>::edge_descriptor edge_descriptor;

    typedef std::pair<unsigned int, unsigned int> Edge;

    Edge* edges = (Edge*)malloc(sizeof(Edge)*graph.edges);
    unsigned int *weight = (unsigned int*)malloc(sizeof(unsigned int)*graph.edges);

    for (int i = 0; i < graph.nodes; ++i)
    {
        for (int j = graph.row_offsets[i]; j < graph.row_offsets[i+1]; ++j)
        {
            edges[j] = Edge(i, graph.column_indices[j]);
            weight[j] = graph.edge_values[j];
        }
    }

    Graph g(edges, edges + graph.edges, weight, graph.nodes);

    std::vector<unsigned int> d(graph.nodes);
    std::vector<vertex_descriptor> p(graph.nodes);
    vertex_descriptor s = vertex(src, g);

    property_map<Graph, vertex_index_t>::type indexmap = get(vertex_index, g);

    //
    // Perform SSSP
    //

    CpuTimer cpu_timer;
    cpu_timer.Start();

    if (MARK_PREDECESSORS)
        dijkstra_shortest_paths(g,
                            s,
                            predecessor_map(boost::make_iterator_property_map(p.begin(), get(boost::vertex_index, g))).
                            distance_map(boost::make_iterator_property_map(d.begin(), get(boost::vertex_index, g))));
    else
        dijkstra_shortest_paths(g,
                            s,
                            distance_map(boost::make_iterator_property_map(d.begin(), get(boost::vertex_index, g))));
    cpu_timer.Stop();
    float elapsed = cpu_timer.ElapsedMillis();

    printf("CPU SSSP finished in %lf msec.\n", elapsed);

    Coo<unsigned int, unsigned int>* sort_dist = NULL;
    Coo<unsigned int, unsigned int>* sort_pred = NULL;
    sort_dist = (Coo<unsigned int, unsigned int>*)malloc(sizeof(Coo<unsigned int, unsigned int>) * graph.nodes);
    if (MARK_PREDECESSORS)
        sort_pred = (Coo<unsigned int, unsigned int>*)malloc(sizeof(Coo<unsigned int, unsigned int>) * graph.nodes);

    graph_traits < Graph >::vertex_iterator vi, vend;
    for (tie(vi, vend) = vertices(g); vi != vend; ++vi)
    {
        sort_dist[(*vi)].row = (*vi);
        sort_dist[(*vi)].col = d[(*vi)];  
    }
    std::stable_sort(sort_dist, sort_dist + graph.nodes, RowFirstTupleCompare<Coo<unsigned int, unsigned int> >);

    if (MARK_PREDECESSORS) {
        for (tie(vi, vend) = vertices(g); vi != vend; ++vi)
        {
            sort_pred[(*vi)].row = (*vi);
            sort_pred[(*vi)].col = p[(*vi)];
        }
        std::stable_sort(sort_pred, sort_pred + graph.nodes, RowFirstTupleCompare<Coo<unsigned int, unsigned int> >);
    }

    for (int i = 0; i < graph.nodes; ++i) {
        node_values[i] = sort_dist[i].col;
    }
    if (MARK_PREDECESSORS)
        for (int i = 0; i < graph.nodes; ++i) {
            node_preds[i] = sort_pred[i].col;
        }

    free(sort_dist);
    if (MARK_PREDECESSORS) free(sort_pred);
}


/**
 * @brief Run SSSP tests
 *
 * @tparam VertexId
 * @tparam Value
 * @tparam SizeT
 * @tparam INSTRUMENT
 * @tparam MARK_PREDECESSORS
 *
 * @param[in] graph Reference to the CSR graph we process on
 * @param[in] src Source node where SSSP starts
 * @param[in] max_grid_size Maximum CTA occupancy
 * @param[in] num_gpus Number of GPUs
 * @param[in] max_queue_sizing Scaling factor used in edge mapping
 *
 */
template <
    typename VertexId,
    typename Value,
    typename SizeT,
    bool INSTRUMENT,
    bool MARK_PREDECESSORS>
void RunTests(
    Csr<VertexId, Value, SizeT> &graph,
    VertexId     src,
    int          max_grid_size,
    float        max_queue_sizing,
    float        max_in_sizing,
    int          num_gpus,
    int          delta_factor,
    int          iterations,
    ContextPtr   *context,
    std::string  partition_method,
    int          *gpu_idx,
    hipStream_t *streams,
    bool         size_check = true)
{
        size_t *org_size = new size_t[num_gpus];

        typedef SSSPProblem<
            VertexId,
            SizeT,
            Value,
            MARK_PREDECESSORS> Problem;

        // Allocate host-side label array (for both reference and gpu-computed results)
        Value     *reference_labels       = new Value[graph.nodes];
        Value     *h_labels               = new Value[graph.nodes];
        Value     *reference_check_label  = (g_quick) ? NULL : reference_labels;
        VertexId  *reference_preds        = NULL;
        VertexId  *h_preds                = NULL;
        VertexId  *reference_check_pred   = NULL;

        if (MARK_PREDECESSORS) {
            reference_preds       = new VertexId[graph.nodes];
            h_preds               = new VertexId[graph.nodes];
            reference_check_pred  = (g_quick) ? NULL : reference_preds;
        }
    
        for (int gpu=0;gpu<num_gpus;gpu++)
        {
            size_t dummy;
            hipSetDevice(gpu_idx[gpu]);
            hipMemGetInfo(&(org_size[gpu]),&dummy);
        }
            
        // Allocate SSSP enactor map
        SSSPEnactor<Problem, INSTRUMENT>* sssp_enactor
            = new SSSPEnactor<Problem, INSTRUMENT>(g_verbose, num_gpus, gpu_idx);

        // Allocate problem on GPU
        Problem *csr_problem = new Problem;
        util::GRError(csr_problem->Init(
            g_stream_from_host,
            graph,
            NULL,
            num_gpus,
            gpu_idx,
            partition_method,
            streams,
            delta_factor,
            max_queue_sizing,
            max_in_sizing), "Problem SSSP Initialization Failed", __FILE__, __LINE__);
        util::GRError(sssp_enactor->Init (context, csr_problem, max_grid_size, size_check), "SSSP Enactor init failed", __FILE__, __LINE__);
        //
        // Compute reference CPU SSSP solution for source-distance
        //
        if (reference_check_label != NULL)
        {
            printf("compute ref value\n");
            SimpleReferenceSssp<VertexId, Value, SizeT, MARK_PREDECESSORS>(
                    graph,
                    reference_check_label,
                    reference_check_pred,
                    src);
            printf("\n");
        }

        Stats      *stats       = new Stats("GPU SSSP");
        long long  total_queued = 0;
        VertexId   search_depth = 0;
        double     avg_duty     = 0.0;
        float      elapsed      = 0.0f;

        // Perform SSSP
        CpuTimer cpu_timer;

        for (int iter = 0; iter < iterations; ++iter)
        {
            util::GRError(csr_problem->Reset(src, sssp_enactor->GetFrontierType(), max_queue_sizing), "SSSP Problem Data Reset Failed", __FILE__, __LINE__); 
            util::GRError(sssp_enactor->Reset(), "SSSP Enactor Reset failed", __FILE__, __LINE__);

            cpu_timer.Start();
            util::GRError(sssp_enactor->Enact(src), "SSSP Problem Enact Failed", __FILE__, __LINE__);
            cpu_timer.Stop();
            elapsed += cpu_timer.ElapsedMillis();
        }
        elapsed /= iterations;

        sssp_enactor->GetStatistics(total_queued, search_depth, avg_duty);

        // Copy out results
        util::GRError(csr_problem->Extract(h_labels, h_preds), "SSSP Problem Data Extraction Failed", __FILE__, __LINE__);

        for (SizeT i=0; i<graph.nodes;i++)
        if (reference_check_label[i]==-1) reference_check_label[i]=util::MaxValue<Value>();

        // Verify the result
        if (reference_check_label != NULL) {
            printf("Label Validity: ");
            int error_num = CompareResults(h_labels, reference_check_label, graph.nodes, true);
            if (error_num > 0)
                printf("%d errors occurred.\n", error_num);
        }
        
        // Display Solution
        printf("\nFirst 40 labels of the GPU result.\n"); 
        DisplaySolution(h_labels, graph.nodes);
        printf("\nFirst 40 labels of the reference CPU result.\n"); 
        DisplaySolution(reference_check_label, graph.nodes);

        if (MARK_PREDECESSORS) {
            printf("\nFirst 40 preds of the GPU result.\n"); 
            DisplaySolution(h_preds, graph.nodes);
            printf("\nFirst 40 preds of the reference CPU result (could be different because the paths are not unique).\n"); 
            DisplaySolution(reference_check_pred, graph.nodes);
        }

        DisplayStats(
            *stats,
            src,
            h_labels,
            graph,
            elapsed,
            search_depth,
            total_queued,
            avg_duty);

        printf("\n\tMemory Usage(B)\t");
        for (int gpu=0;gpu<num_gpus;gpu++)
        if (num_gpus>1) {if (gpu!=0) printf(" #keys%d\t #ins%d,0\t #ins%d,1",gpu,gpu,gpu); else printf(" $keys%d", gpu);}
        else printf(" #keys%d", gpu);
        if (num_gpus>1) printf(" #keys%d",num_gpus);
        printf("\n");
        double max_key_sizing=0, max_in_sizing_=0;
        for (int gpu=0;gpu<num_gpus;gpu++)
        {
            size_t gpu_free,dummy;
            hipSetDevice(gpu_idx[gpu]);
            hipMemGetInfo(&gpu_free,&dummy);
            printf("GPU_%d\t %ld",gpu_idx[gpu],org_size[gpu]-gpu_free);
            for (int i=0;i<num_gpus;i++)
            {
                SizeT x=csr_problem->graph_slices[gpu]->frontier_queues[i].keys[0].GetSize();
                printf("\t %d", x);
                double factor = 1.0*x/(num_gpus>1?csr_problem->graph_slices[gpu]->in_counter[i]:csr_problem->graph_slices[gpu]->nodes);
                if (factor > max_key_sizing) max_key_sizing=factor;
                if (num_gpus>1 && i!=0 )
                for (int t=0;t<2;t++)
                {
                    x=csr_problem->data_slices[gpu][0].keys_in[t][i].GetSize();
                    printf("\t %d", x);
                    factor = 1.0*x/csr_problem->graph_slices[gpu]->in_counter[i];
                    if (factor > max_in_sizing_) max_in_sizing_=factor;
                }
            }
            if (num_gpus>1) printf("\t %d",csr_problem->graph_slices[gpu]->frontier_queues[num_gpus].keys[0].GetSize());
            printf("\n");
        }
        printf("\t key_sizing =\t %lf", max_key_sizing);
        if (num_gpus>1) printf("\t in_sizing =\t %lf", max_in_sizing_);
        printf("\n");

        // Cleanup
        if (org_size        ) {delete[] org_size        ; org_size         = NULL;}
        if (stats           ) {delete   stats           ; stats            = NULL;}
        if (sssp_enactor    ) {delete   sssp_enactor    ; sssp_enactor     = NULL;}
        if (csr_problem     ) {delete   csr_problem     ; csr_problem      = NULL;}
        if (reference_labels) {delete[] reference_labels; reference_labels = NULL;}
        if (h_labels        ) {delete[] h_labels        ; h_labels         = NULL;}
        if (reference_preds ) {delete[] reference_preds ; reference_preds  = NULL;}
        if (h_preds         ) {delete[] h_preds         ; h_preds          = NULL;}

        //hipDeviceSynchronize();
}

/**
 * @brief RunTests entry
 *
 * @tparam VertexId
 * @tparam Value
 * @tparam SizeT
 *
 * @param[in] graph Reference to the CSR graph we process on
 * @param[in] args Reference to the command line arguments
 */
template <
    typename VertexId,
    typename Value,
    typename SizeT>
void RunTests(
    Csr<VertexId, Value, SizeT> &graph,
    CommandLineArgs             &args,
    int                         num_gpus,
    ContextPtr                  *context,
    int                         *gpu_idx,
    hipStream_t                *streams)
{
    VertexId            src                 = -1;           // Use whatever the specified graph-type's default is
    std::string         src_str;
    bool                instrumented        = false;        // Whether or not to collect instrumentation from kernels
    int                 max_grid_size       = 0;            // maximum grid size (0: leave it up to the enactor)
    //int                 num_gpus            = 1;            // Number of GPUs for multi-gpu enactor to use
    float               max_queue_sizing    = 1.0;
    float               max_in_sizing       = 1.0;
    bool                mark_pred           = false;
    std::string         partition_method    = "random";
    int                 iterations          = 1;
    bool                disable_size_check  = false;
    int                 delta_factor        = 16;

    instrumented = args.CheckCmdLineFlag("instrumented");
    disable_size_check = args.CheckCmdLineFlag("disable-size-check");
    args.GetCmdLineArgument("src", src_str);
    if (src_str.empty()) {
        src = 0;
    } else if (src_str.compare("randomize") == 0) {
        src = graphio::RandomNode(graph.nodes);
    } else if (src_str.compare("largestdegree") == 0) {
        int temp;
        src = graph.GetNodeWithHighestDegree(temp);
    } else {
        args.GetCmdLineArgument("src", src);
    }

    g_verbose = args.CheckCmdLineFlag("v"        );
    g_quick   = args.CheckCmdLineFlag("quick"    );
    mark_pred = args.CheckCmdLineFlag("mark-pred");
    args.GetCmdLineArgument("iteration-num", iterations      );
    args.GetCmdLineArgument("queue-sizing",  max_queue_sizing);
    args.GetCmdLineArgument("in-sizing",     max_in_sizing   );
    args.GetCmdLineArgument("grid-size",     max_grid_size   );
    args.GetCmdLineArgument("delta-factor",  delta_factor    );
    if (args.CheckCmdLineFlag  ("partition_method")) 
        args.GetCmdLineArgument("partition_method",partition_method);

    if (mark_pred) {
        if (instrumented) {
            RunTests<VertexId, Value, SizeT, true, true>(
                    graph,
                    src,
                    max_grid_size,
                    max_queue_sizing,
                    max_in_sizing,
                    num_gpus,
                    delta_factor,
                    iterations,
                    context,
                    partition_method,
                    gpu_idx,
                    streams,
                    !disable_size_check);
        } else {
            RunTests<VertexId, Value, SizeT, false, true>(
                    graph,
                    src,
                    max_grid_size,
                    max_queue_sizing,
                    max_in_sizing,
                    num_gpus,
                    delta_factor,
                    iterations,
                    context,
                    partition_method,
                    gpu_idx,
                    streams,
                    !disable_size_check);
        }
    } else {
        if (instrumented) {
            RunTests<VertexId, Value, SizeT, true, false>(
                    graph,
                    src,
                    max_grid_size,
                    max_queue_sizing,
                    max_in_sizing,
                    num_gpus,
                    delta_factor,
                    iterations,
                    context,
                    partition_method,
                    gpu_idx,
                    streams,
                    !disable_size_check);
        } else {
            RunTests<VertexId, Value, SizeT, false, false>(
                    graph,
                    src,
                    max_grid_size,
                    max_queue_sizing,
                    max_in_sizing,
                    num_gpus,
                    delta_factor,
                    iterations,
                    context,
                    partition_method,
                    gpu_idx,
                    streams,
                    !disable_size_check);
        }
    }

}



/******************************************************************************
* Main
******************************************************************************/

int cpp_main( int argc, char** argv)
{
    CommandLineArgs args(argc, argv);
    int          num_gpus = 0;
    int          *gpu_idx = NULL;
    ContextPtr   *context = NULL;
    hipStream_t *streams = NULL;

    if ((argc < 2) || (args.CheckCmdLineFlag("help"))) {
        Usage();
        return 1;
    }

    if (args.CheckCmdLineFlag  ("device"))
    {
        std::vector<int> gpus;
        args.GetCmdLineArguments<int>("device",gpus);
        num_gpus   = gpus.size();
        gpu_idx    = new int[num_gpus];
        for (int i=0;i<num_gpus;i++)
            gpu_idx[i] = gpus[i];
    } else {
        num_gpus   = 1;
        gpu_idx    = new int[num_gpus];
        gpu_idx[0] = 0;
    }
    streams  = new hipStream_t[num_gpus * num_gpus *2];
    context  = new ContextPtr  [num_gpus * num_gpus];
    printf("Using %d gpus: ", num_gpus);
    for (int gpu=0;gpu<num_gpus;gpu++)
    {
        printf(" %d ", gpu_idx[gpu]);
        util::SetDevice(gpu_idx[gpu]);
        for (int i=0;i<num_gpus*2;i++)
        {
            int _i=gpu*num_gpus*2+i;
            util::GRError(hipStreamCreate(&streams[_i]), "hipStreamCreate fialed.",__FILE__,__LINE__);
            if (i<num_gpus) context[gpu*num_gpus+i] = mgpu::CreateCudaDeviceAttachStream(gpu_idx[gpu],streams[_i]);
        }
    }
    printf("\n"); fflush(stdout);
    
    // Parse graph-contruction params
    g_undirected = args.CheckCmdLineFlag("undirected");
    std::string graph_type = argv[1];
    int flags = args.ParsedArgc();
    int graph_args = argc - flags - 1;

    if (graph_args < 1) {
        Usage();
        return 1;
    }
	
    //
    // Construct graph and perform search(es)
    //

    if (graph_type == "market") {
    // Matrix-market coordinate-formatted graph file

        typedef int VertexId;                   // Use as the node identifier type
        typedef int Value;             // Use as the value type
        typedef int SizeT;                      // Use as the graph size type
        Csr<VertexId, Value, SizeT> csr(false); // default value for stream_from_host is false

        if (graph_args < 1) { Usage(); return 1; }
        char *market_filename = (graph_args == 2) ? argv[2] : NULL;
        if (graphio::BuildMarketGraph<true>(
            market_filename, 
            csr, 
            g_undirected,
            false) != 0) // no inverse graph
        {
            return 1;
        }

        csr.PrintHistogram();
        csr.DisplayGraph(true); //print graph with edge_value
        
        csr.GetAverageEdgeValue();
        csr.GetAverageDegree();
        int max_degree;
        csr.GetNodeWithHighestDegree(max_degree);
        printf("max degree:%d\n", max_degree);
	
        // Run tests
        RunTests(csr, args, num_gpus, context, gpu_idx, streams);

    } else {
        // Unknown graph type
        fprintf(stderr, "Unspecified graph type\n");
        return 1;
    }

    return 0;
}
