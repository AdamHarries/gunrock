#include "hip/hip_runtime.h"
// ----------------------------------------------------------------
// Gunrock -- Fast and Efficient GPU Graph Library
// ----------------------------------------------------------------
// This source code is distributed under the terms of LICENSE.TXT
// in the root directory of this source distribution.
// ----------------------------------------------------------------

/**
 * @file
 * test_sssp.cu
 *
 * @brief Simple test driver program for single source shorest path.
 */

#include <stdio.h> 
#include <string>
#include <deque>
#include <vector>
#include <iostream>

// Utilities and correctness-checking
#include <gunrock/util/test_utils.cuh>

// Graph construction utils
#include <gunrock/graphio/market.cuh>

// SSSP includes
#include <gunrock/app/sssp/sssp_enactor.cuh>
#include <gunrock/app/sssp/sssp_problem.cuh>
#include <gunrock/app/sssp/sssp_functor.cuh>

// Operator includes
#include <gunrock/oprtr/advance/kernel.cuh>
#include <gunrock/oprtr/filter/kernel.cuh>
#include <gunrock/priority_queue/kernel.cuh>

#include <moderngpu.cuh>

// Boost includes for CPU dijkstra SSSP reference algorithms
#include <boost/config.hpp>
#include <boost/graph/graph_traits.hpp>
#include <boost/graph/adjacency_list.hpp>
#include <boost/graph/dijkstra_shortest_paths.hpp>
#include <boost/property_map/property_map.hpp>

using namespace gunrock;
using namespace gunrock::util;
using namespace gunrock::oprtr;
using namespace gunrock::app::sssp;


/******************************************************************************
 * Defines, constants, globals 
 ******************************************************************************/

//bool g_verbose;
//bool g_undirected;
//bool g_quick;
//bool g_stream_from_host;

/******************************************************************************
 * Housekeeping Routines
 ******************************************************************************/
 void Usage()
 {
 printf("\ntest_sssp <graph type> <graph type args> [--device=<device_index>] "
        "[--undirected] [--instrumented] [--src=<source index>] [--quick]\n"
        "[--v] [mark-pred] [--queue-sizing=<scale factor>] "
        "[--in-sizing=<in/out queue scale factor>] [--disable-size-check] "
        "[--grid-size=<grid size>] [partition_method=random / biasrandom / clustered / metis]\n"
        "\n"
        "Graph types and args:\n"
        "  market [<file>]\n"
        "    Reads a Matrix-Market coordinate-formatted graph of directed/undirected\n"
        "    edges from stdin (or from the optionally-specified file).\n"
        "  --device=<device_index>  Set GPU device for running the graph primitive.\n"
        "  --undirected If set then treat the graph as undirected.\n"
        "  --instrumented If set then kernels keep track of queue-search_depth\n"
        "  and barrier duty (a relative indicator of load imbalance.)\n"
        "  --src Begins SSSP from the vertex <source index>. If set as randomize\n"
        "  then will begin with a random source vertex.\n"
        "  If set as largestdegree then will begin with the node which has\n"
        "  largest degree.\n"
        "  --quick If set will skip the CPU validation code.\n"
        "  --v Whether to show debug info.\n"
        "  --mark-pred If set then keep not only label info but also predecessor info.\n"
        "  --queue-sizing Allocates a frontier queue sized at (graph-edges * <scale factor>).\n"
        );
 }

 /**
  * @brief Displays the SSSP result (i.e., distance from source)
  *
  * @param[in] source_path Search depth from the source for each node.
  * @param[in] preds Predecessor node id for each node.
  * @param[in] nodes Number of nodes in the graph.
  * @param[in] MARK_PREDECESSORS Whether to show predecessor of each node.
  */
 template<typename VertexId, typename SizeT>
 void DisplaySolution(VertexId *source_path, SizeT nodes)
 {
    if (nodes > 40)
        nodes = 40;
    printf("[");
    for (VertexId i = 0; i < nodes; ++i) {
        PrintValue(i);
        printf(":");
        PrintValue(source_path[i]);
        printf(" ");
    }
    printf("]\n");
 }

 /**
  * Performance/Evaluation statistics
  */ 

struct Stats {
    const char *name;
    Statistic rate;
    Statistic search_depth;
    Statistic redundant_work;
    Statistic duty;

    Stats() : name(NULL), rate(), search_depth(), redundant_work(), duty() {}
    Stats(const char *name) : name(name), rate(), search_depth(), redundant_work(), duty() {}
};

struct Test_Parameter : gunrock::app::TestParameter_Base {
public:
    //bool          mark_predecessors ;// Whether or not to mark src-distance vs. parent vertices
    int delta_factor;

    Test_Parameter()
    { 
        delta_factor = 16;
        mark_predecessors = false;
    }   

    ~Test_Parameter()
    {   
    }   

    void Init(CommandLineArgs &args)
    {   
        TestParameter_Base::Init(args);
        mark_predecessors = args.CheckCmdLineFlag("mark-pred");
        args.GetCmdLineArgument("delta-factor"    , delta_factor    );
    }   
};

/**
 * @brief Displays timing and correctness statistics
 *
 * @tparam MARK_PREDECESSORS
 * @tparam VertexId
 * @tparam Value
 * @tparam SizeT
 * 
 * @param[in] stats Reference to the Stats object defined in RunTests
 * @param[in] src Source node where SSSP starts
 * @param[in] h_labels Host-side vector stores computed labels for validation
 * @param[in] graph Reference to the CSR graph we process on
 * @param[in] elapsed Total elapsed kernel running time
 * @param[in] search_depth Maximum search depth of the SSSP algorithm
 * @param[in] total_queued Total element queued in SSSP kernel running process
 * @param[in] avg_duty Average duty of the SSSP kernels
 */
template<
    typename VertexId,
    typename Value,
    typename SizeT>
void DisplayStats(
    Stats               &stats,
    VertexId            src,
    Value               *h_labels,
    const Csr<VertexId, Value, SizeT> &graph,
    double              elapsed,
    VertexId            search_depth,
    long long           total_queued,
    double              avg_duty)
{
    // Compute nodes and edges visited
    SizeT edges_visited = 0;
    SizeT nodes_visited = 0;
    for (VertexId i = 0; i < graph.nodes; ++i) {
        if (h_labels[i] < util::MaxValue<VertexId>()) {
            ++nodes_visited;
            edges_visited += graph.row_offsets[i+1] - graph.row_offsets[i];
        }
    }

    double redundant_work = 0.0;
    if (total_queued > 0) {
        redundant_work = ((double) total_queued - edges_visited) / edges_visited;        // measure duplicate edges put through queue
    }
    redundant_work *= 100;

    // Display test name
    printf("[%s] finished. ", stats.name);

    // Display statistics
    if (nodes_visited < 5) {
        printf("Fewer than 5 vertices visited.\n");
    } else {
        // Display the specific sample statistics
        double m_teps = (double) edges_visited / (elapsed * 1000.0);
        printf(" elapsed: %.3f ms, rate: %.3f MiEdges/s", elapsed, m_teps);
        if (search_depth != 0) printf(", search_depth: %lld", (long long) search_depth);
        if (avg_duty != 0) {
            printf("\n avg CTA duty: %.2f%%", avg_duty * 100);
        }
        printf("\n src: %lld, nodes_visited: %lld, edges visited: %lld",
            (long long) src, (long long) nodes_visited, (long long) edges_visited);
        if (total_queued > 0) {
            printf(", total queued: %lld", total_queued);
        }
        if (redundant_work > 0) {
            printf(", redundant work: %.2f%%", redundant_work);
        }
        printf("\n");
    }
}

/******************************************************************************
 * SSSP Testing Routines
 *****************************************************************************/

 /**
  * @brief A simple CPU-based reference SSSP ranking implementation.
  *
  * @tparam VertexId
  * @tparam Value
  * @tparam SizeT
  * @tparam MARK_PREDECESSORS
  *
  * @param[in] graph Reference to the CSR graph we process on
  * @param[in] node_values Host-side vector to store CPU computed labels for each node
  * @param[in] node_preds Host-side vector to store CPU computed predecessors for each node
  * @param[in] src Source node where SSSP starts
  */
template<
    typename VertexId,
    typename Value,
    typename SizeT,
    bool     MARK_PREDECESSORS>
void SimpleReferenceSssp(
    const Csr<VertexId, Value, SizeT>       &graph,
    Value                                   *node_values,
    VertexId                                *node_preds,
    VertexId                                src)
{
    using namespace boost; 
    // Prepare Boost Datatype and Data structure
    typedef adjacency_list<vecS, vecS, directedS,
            no_property, property <edge_weight_t, unsigned int> > Graph;
    typedef graph_traits<Graph>::vertex_descriptor vertex_descriptor;
    typedef graph_traits<Graph>::edge_descriptor edge_descriptor;

    typedef std::pair<unsigned int, unsigned int> Edge;

    Edge* edges = (Edge*)malloc(sizeof(Edge)*graph.edges);
    unsigned int *weight = (unsigned int*)malloc(sizeof(unsigned int)*graph.edges);

    for (int i = 0; i < graph.nodes; ++i)
    {
        for (int j = graph.row_offsets[i]; j < graph.row_offsets[i+1]; ++j)
        {
            edges[j] = Edge(i, graph.column_indices[j]);
            weight[j] = graph.edge_values[j];
        }
    }

    Graph g(edges, edges + graph.edges, weight, graph.nodes);

    std::vector<unsigned int> d(graph.nodes);
    std::vector<vertex_descriptor> p(graph.nodes);
    vertex_descriptor s = vertex(src, g);

    property_map<Graph, vertex_index_t>::type indexmap = get(vertex_index, g);

    //
    // Perform SSSP
    //

    CpuTimer cpu_timer;
    cpu_timer.Start();

    if (MARK_PREDECESSORS)
        dijkstra_shortest_paths(g,
                            s,
                            predecessor_map(boost::make_iterator_property_map(p.begin(), get(boost::vertex_index, g))).
                            distance_map(boost::make_iterator_property_map(d.begin(), get(boost::vertex_index, g))));
    else
        dijkstra_shortest_paths(g,
                            s,
                            distance_map(boost::make_iterator_property_map(d.begin(), get(boost::vertex_index, g))));
    cpu_timer.Stop();
    float elapsed = cpu_timer.ElapsedMillis();

    printf("CPU SSSP finished in %lf msec.\n", elapsed);

    Coo<unsigned int, unsigned int>* sort_dist = NULL;
    Coo<unsigned int, unsigned int>* sort_pred = NULL;
    sort_dist = (Coo<unsigned int, unsigned int>*)malloc(sizeof(Coo<unsigned int, unsigned int>) * graph.nodes);
    if (MARK_PREDECESSORS)
        sort_pred = (Coo<unsigned int, unsigned int>*)malloc(sizeof(Coo<unsigned int, unsigned int>) * graph.nodes);

    graph_traits < Graph >::vertex_iterator vi, vend;
    for (tie(vi, vend) = vertices(g); vi != vend; ++vi)
    {
        sort_dist[(*vi)].row = (*vi);
        sort_dist[(*vi)].col = d[(*vi)];  
    }
    std::stable_sort(sort_dist, sort_dist + graph.nodes, RowFirstTupleCompare<Coo<unsigned int, unsigned int> >);

    if (MARK_PREDECESSORS) {
        for (tie(vi, vend) = vertices(g); vi != vend; ++vi)
        {
            sort_pred[(*vi)].row = (*vi);
            sort_pred[(*vi)].col = p[(*vi)];
        }
        std::stable_sort(sort_pred, sort_pred + graph.nodes, RowFirstTupleCompare<Coo<unsigned int, unsigned int> >);
    }

    for (int i = 0; i < graph.nodes; ++i) {
        node_values[i] = sort_dist[i].col;
    }
    if (MARK_PREDECESSORS)
        for (int i = 0; i < graph.nodes; ++i) {
            node_preds[i] = sort_pred[i].col;
        }

    free(sort_dist);
    if (MARK_PREDECESSORS) free(sort_pred);
}


/**
 * @brief Run SSSP tests
 *
 * @tparam VertexId
 * @tparam Value
 * @tparam SizeT
 * @tparam INSTRUMENT
 * @tparam MARK_PREDECESSORS
 *
 * @param[in] graph Reference to the CSR graph we process on
 * @param[in] src Source node where SSSP starts
 * @param[in] max_grid_size Maximum CTA occupancy
 * @param[in] num_gpus Number of GPUs
 * @param[in] max_queue_sizing Scaling factor used in edge mapping
 *
 */
template <
    typename VertexId,
    typename Value,
    typename SizeT,
    bool INSTRUMENT,
    bool DEBUG,
    bool SIZE_CHECK,
    bool MARK_PREDECESSORS>
void RunTests(Test_Parameter *parameter)
{
    typedef SSSPProblem<
        VertexId,
        SizeT,
        Value,
        MARK_PREDECESSORS> Problem;

    typedef SSSPEnactor<
        Problem,
        INSTRUMENT,
        DEBUG,
        SIZE_CHECK> Enactor;

    Csr<VertexId, Value, SizeT>
                 *graph                 = (Csr<VertexId, Value, SizeT>*)parameter->graph;
    VertexId      src                   = (VertexId)parameter -> src;
    int           max_grid_size         = parameter -> max_grid_size;
    int           num_gpus              = parameter -> num_gpus;
    double        max_queue_sizing      = parameter -> max_queue_sizing;
    double        max_in_sizing         = parameter -> max_in_sizing;
    ContextPtr   *context               = (ContextPtr*)parameter -> context;
    std::string   partition_method      = parameter -> partition_method;
    int          *gpu_idx               = parameter -> gpu_idx;
    hipStream_t *streams               = parameter -> streams;
    float         partition_factor      = parameter -> partition_factor;
    int           partition_seed        = parameter -> partition_seed;
    bool          g_quick               = parameter -> g_quick;
    bool          g_stream_from_host    = parameter -> g_stream_from_host;
    int           delta_factor          = parameter -> delta_factor;
    int           iterations            = parameter -> iterations;
    size_t       *org_size              = new size_t[num_gpus];
    // Allocate host-side label array (for both reference and gpu-computed results)
    Value        *reference_labels      = new Value[graph->nodes];
    Value        *h_labels              = new Value[graph->nodes];
    Value        *reference_check_label = (g_quick) ? NULL : reference_labels;
    VertexId     *reference_preds       = MARK_PREDECESSORS ? new VertexId[graph->nodes] : NULL;
    VertexId     *h_preds               = MARK_PREDECESSORS ? new VertexId[graph->nodes] : NULL;
    VertexId     *reference_check_pred  = (g_quick || !MARK_PREDECESSORS) ? NULL : reference_preds;

    for (int gpu=0;gpu<num_gpus;gpu++)
    {
        size_t dummy;
        hipSetDevice(gpu_idx[gpu]);
        hipMemGetInfo(&(org_size[gpu]),&dummy);
    }
        
    // Allocate SSSP enactor map
    Enactor* enactor = new Enactor(num_gpus, gpu_idx);

    // Allocate problem on GPU
    Problem *problem = new Problem;
    util::GRError(problem->Init(
        g_stream_from_host,
        graph,
        NULL,
        num_gpus,
        gpu_idx,
        partition_method,
        streams,
        delta_factor,
        max_queue_sizing,
        max_in_sizing,
        partition_factor,
        partition_seed), "Problem SSSP Initialization Failed", __FILE__, __LINE__);
    util::GRError(enactor->Init (context, problem, max_grid_size), "SSSP Enactor init failed", __FILE__, __LINE__);
    //
    // Compute reference CPU SSSP solution for source-distance
    //
    if (reference_check_label != NULL)
    {
        printf("compute ref value\n");
        SimpleReferenceSssp<VertexId, Value, SizeT, MARK_PREDECESSORS>(
                *graph,
                reference_check_label,
                reference_check_pred,
                src);
        printf("\n");
    }

    Stats      *stats       = new Stats("GPU SSSP");
    long long  total_queued = 0;
    VertexId   search_depth = 0;
    double     avg_duty     = 0.0;
    float      elapsed      = 0.0f;

    // Perform SSSP
    CpuTimer cpu_timer;

    for (int iter = 0; iter < iterations; ++iter)
    {
        util::GRError(problem->Reset(src, enactor->GetFrontierType(), max_queue_sizing), "SSSP Problem Data Reset Failed", __FILE__, __LINE__); 
        util::GRError(enactor->Reset(), "SSSP Enactor Reset failed", __FILE__, __LINE__);

        printf("__________________________\n");fflush(stdout);
        cpu_timer.Start();
        util::GRError(enactor->Enact(src), "SSSP Problem Enact Failed", __FILE__, __LINE__);
        cpu_timer.Stop();
        printf("--------------------------\n");fflush(stdout);
        elapsed += cpu_timer.ElapsedMillis();
    }
    elapsed /= iterations;

    enactor->GetStatistics(total_queued, search_depth, avg_duty);

    // Copy out results
    util::GRError(problem->Extract(h_labels, h_preds), "SSSP Problem Data Extraction Failed", __FILE__, __LINE__);

    for (SizeT i=0; i<graph->nodes;i++)
    if (reference_check_label[i]==-1) reference_check_label[i]=util::MaxValue<Value>();

    // Verify the result
    if (reference_check_label != NULL) {
        printf("Label Validity: ");
        int error_num = CompareResults(h_labels, reference_check_label, graph->nodes, true);
        if (error_num > 0)
            printf("%d errors occurred.\n", error_num);
    }
    
    // Display Solution
    printf("\nFirst 40 labels of the GPU result.\n"); 
    DisplaySolution(h_labels, graph->nodes);
    printf("\nFirst 40 labels of the reference CPU result.\n"); 
    DisplaySolution(reference_check_label, graph->nodes);

    if (MARK_PREDECESSORS) {
        printf("\nFirst 40 preds of the GPU result.\n"); 
        DisplaySolution(h_preds, graph->nodes);
        printf("\nFirst 40 preds of the reference CPU result (could be different because the paths are not unique).\n"); 
        DisplaySolution(reference_check_pred, graph->nodes);
    }

    DisplayStats(
        *stats,
        src,
        h_labels,
        *graph,
        elapsed,
        search_depth,
        total_queued,
        avg_duty);

    printf("\n\tMemory Usage(B)\t");
    for (int gpu=0;gpu<num_gpus;gpu++)
    if (num_gpus>1) {if (gpu!=0) printf(" #keys%d\t #ins%d,0\t #ins%d,1",gpu,gpu,gpu); else printf(" $keys%d", gpu);}
    else printf(" #keys%d", gpu);
    if (num_gpus>1) printf(" #keys%d",num_gpus);
    printf("\n");
    double max_key_sizing=0, max_in_sizing_=0;
    for (int gpu=0;gpu<num_gpus;gpu++)
    {
        size_t gpu_free,dummy;
        hipSetDevice(gpu_idx[gpu]);
        hipMemGetInfo(&gpu_free,&dummy);
        printf("GPU_%d\t %ld",gpu_idx[gpu],org_size[gpu]-gpu_free);
        for (int i=0;i<num_gpus;i++)
        {
            SizeT x=problem->data_slices[gpu]->frontier_queues[i].keys[0].GetSize();
            printf("\t %d", x);
            double factor = 1.0*x/(num_gpus>1?problem->graph_slices[gpu]->in_counter[i]:problem->graph_slices[gpu]->nodes);
            if (factor > max_key_sizing) max_key_sizing=factor;
            if (num_gpus>1 && i!=0 )
            for (int t=0;t<2;t++)
            {
                x=problem->data_slices[gpu][0].keys_in[t][i].GetSize();
                printf("\t %d", x);
                factor = 1.0*x/problem->graph_slices[gpu]->in_counter[i];
                if (factor > max_in_sizing_) max_in_sizing_=factor;
            }
        }
        if (num_gpus>1) printf("\t %d",problem->data_slices[gpu]->frontier_queues[num_gpus].keys[0].GetSize());
        printf("\n");
    }
    printf("\t key_sizing =\t %lf", max_key_sizing);
    if (num_gpus>1) printf("\t in_sizing =\t %lf", max_in_sizing_);
    printf("\n");

    // Cleanup
    if (org_size        ) {delete[] org_size        ; org_size         = NULL;}
    if (stats           ) {delete   stats           ; stats            = NULL;}
    if (enactor         ) {delete   enactor         ; enactor          = NULL;}
    if (problem         ) {delete   problem         ; problem          = NULL;}
    if (reference_labels) {delete[] reference_labels; reference_labels = NULL;}
    if (h_labels        ) {delete[] h_labels        ; h_labels         = NULL;}
    if (reference_preds ) {delete[] reference_preds ; reference_preds  = NULL;}
    if (h_preds         ) {delete[] h_preds         ; h_preds          = NULL;}

    //hipDeviceSynchronize();
}

template <
    typename    VertexId,
    typename    Value,
    typename    SizeT,
    bool        INSTRUMENT,
    bool        DEBUG,
    bool        SIZE_CHECK>
void RunTests_mark_predecessors(Test_Parameter *parameter)
{
    if (parameter->mark_predecessors) RunTests
        <VertexId, Value, SizeT, INSTRUMENT, DEBUG, SIZE_CHECK,
        true > (parameter);
   else RunTests
        <VertexId, Value, SizeT, INSTRUMENT, DEBUG, SIZE_CHECK,
        false> (parameter);
}

template <
    typename      VertexId,
    typename      Value,
    typename      SizeT,
    bool          INSTRUMENT,
    bool          DEBUG>
void RunTests_size_check(Test_Parameter *parameter)
{
    if (parameter->size_check) RunTests_mark_predecessors
        <VertexId, Value, SizeT, INSTRUMENT, DEBUG,
        true > (parameter);
   else RunTests_mark_predecessors
        <VertexId, Value, SizeT, INSTRUMENT, DEBUG,
        false> (parameter);
}

template <
    typename    VertexId,
    typename    Value,
    typename    SizeT,
    bool        INSTRUMENT>
void RunTests_debug(Test_Parameter *parameter)
{
    if (parameter->debug) RunTests_size_check
        <VertexId, Value, SizeT, INSTRUMENT,
        true > (parameter);
    else RunTests_size_check
        <VertexId, Value, SizeT, INSTRUMENT,
        false> (parameter);
}

template <
    typename      VertexId,
    typename      Value,
    typename      SizeT>
void RunTests_instrumented(Test_Parameter *parameter)
{
    if (parameter->instrumented) RunTests_debug
        <VertexId, Value, SizeT,
        true > (parameter);
    else RunTests_debug
        <VertexId, Value, SizeT,
        false> (parameter);
}

/**
 * @brief RunTests entry
 *
 * @tparam VertexId
 * @tparam Value
 * @tparam SizeT
 *
 * @param[in] graph Reference to the CSR graph we process on
 * @param[in] args Reference to the command line arguments
 */
template <
    typename VertexId,
    typename Value,
    typename SizeT>
void RunTests(
    Csr<VertexId, Value, SizeT> *graph,
    CommandLineArgs             &args,
    int                         num_gpus,
    ContextPtr                  *context,
    int                         *gpu_idx,
    hipStream_t                *streams)
{
    string src_str = "";
    Test_Parameter *parameter = new Test_Parameter;
    
    parameter -> Init(args);
    parameter -> graph              = graph;
    parameter -> num_gpus           = num_gpus;
    parameter -> context            = context;
    parameter -> gpu_idx            = gpu_idx;
    parameter -> streams            = streams;

    args.GetCmdLineArgument("src", src_str);
    if (src_str.empty()) {
        parameter->src = 0;
    } else if (src_str.compare("randomize") == 0) {
        parameter->src = graphio::RandomNode(graph->nodes);
    } else if (src_str.compare("largestdegree") == 0) {
        int temp;
        parameter->src = graph->GetNodeWithHighestDegree(temp);
    } else {
        args.GetCmdLineArgument("src", parameter->src);
    }
    printf("src = %lld\n", parameter->src);

    RunTests_instrumented<VertexId, Value, SizeT>(parameter);
}



/******************************************************************************
* Main
******************************************************************************/

int cpp_main( int argc, char** argv)
{
    CommandLineArgs args(argc, argv);
    int          num_gpus = 0;
    int          *gpu_idx = NULL;
    ContextPtr   *context = NULL;
    hipStream_t *streams = NULL;
    bool          g_undirected = false;

    if ((argc < 2) || (args.CheckCmdLineFlag("help"))) {
        Usage();
        return 1;
    }

    if (args.CheckCmdLineFlag  ("device"))
    {
        std::vector<int> gpus;
        args.GetCmdLineArguments<int>("device",gpus);
        num_gpus   = gpus.size();
        gpu_idx    = new int[num_gpus];
        for (int i=0;i<num_gpus;i++)
            gpu_idx[i] = gpus[i];
    } else {
        num_gpus   = 1;
        gpu_idx    = new int[num_gpus];
        gpu_idx[0] = 0;
    }
    streams  = new hipStream_t[num_gpus * num_gpus *2];
    context  = new ContextPtr  [num_gpus * num_gpus];
    printf("Using %d gpus: ", num_gpus);
    for (int gpu=0;gpu<num_gpus;gpu++)
    {
        printf(" %d ", gpu_idx[gpu]);
        util::SetDevice(gpu_idx[gpu]);
        for (int i=0;i<num_gpus*2;i++)
        {
            int _i=gpu*num_gpus*2+i;
            util::GRError(hipStreamCreate(&streams[_i]), "hipStreamCreate fialed.",__FILE__,__LINE__);
            if (i<num_gpus) context[gpu*num_gpus+i] = mgpu::CreateCudaDeviceAttachStream(gpu_idx[gpu],streams[_i]);
        }
    }
    printf("\n"); fflush(stdout);
    
    // Parse graph-contruction params
    g_undirected = args.CheckCmdLineFlag("undirected");
    std::string graph_type = argv[1];
    int flags = args.ParsedArgc();
    int graph_args = argc - flags - 1;

    if (graph_args < 1) {
        Usage();
        return 1;
    }
	
    //
    // Construct graph and perform search(es)
    //

    if (graph_type == "market") {
    // Matrix-market coordinate-formatted graph file

        typedef int VertexId;                   // Use as the node identifier type
        typedef int Value;             // Use as the value type
        typedef int SizeT;                      // Use as the graph size type
        Csr<VertexId, Value, SizeT> csr(false); // default value for stream_from_host is false

        if (graph_args < 1) { Usage(); return 1; }
        char *market_filename = (graph_args == 2) ? argv[2] : NULL;
        if (graphio::BuildMarketGraph<true>(
            market_filename, 
            csr, 
            g_undirected,
            false) != 0) // no inverse graph
        {
            return 1;
        }

        csr.PrintHistogram();
        csr.DisplayGraph(true); //print graph with edge_value
        
        csr.GetAverageEdgeValue();
        csr.GetAverageDegree();
        int max_degree;
        csr.GetNodeWithHighestDegree(max_degree);
        printf("max degree:%d\n", max_degree);
	
        // Run tests
        RunTests(&csr, args, num_gpus, context, gpu_idx, streams);

    } else {
        // Unknown graph type
        fprintf(stderr, "Unspecified graph type\n");
        return 1;
    }

    return 0;
}
