// ----------------------------------------------------------------
// Gunrock -- Fast and Efficient GPU Graph Library
// ----------------------------------------------------------------
// This source code is distributed under the terms of LICENSE.TXT
// in the root directory of this source distribution.
// ----------------------------------------------------------------

/**
 * @file
 * test_bfs.cu
 *
 * @brief Simple test driver program for breadth-first search.
 */

#include <stdio.h> 
#include <string>
#include <deque>
#include <vector>
#include <iostream>

// Utilities and correctness-checking
#include <gunrock/util/test_utils.cuh>

// Graph construction utils
#include <gunrock/graphio/market.cuh>

// BFS includes
#include <gunrock/app/bfs/bfs_enactor.cuh>
#include <gunrock/app/bfs/bfs_problem.cuh>
#include <gunrock/app/bfs/bfs_functor.cuh>

// Operator includes
#include <gunrock/oprtr/advance/kernel.cuh>
#include <gunrock/oprtr/filter/kernel.cuh>

#include <moderngpu.cuh>

using namespace gunrock;
using namespace gunrock::util;
using namespace gunrock::oprtr;
using namespace gunrock::app::bfs;


/******************************************************************************
 * Defines, constants, globals 
 ******************************************************************************/

bool g_verbose;
bool g_undirected;
bool g_quick;
bool g_stream_from_host;

/******************************************************************************
 * Housekeeping Routines
 ******************************************************************************/
 void Usage()
 {
 printf("\ntest_bfs <graph type> <graph type args> [--device=<device_index>] "
        "[--undirected] [--instrumented] [--src=<source index>] [--quick] "
        "[--mark-pred] [--queue-sizing=<scale factor>] "
        "[--in-sizing=<in/out queue scale factor>] [--disable-size-check] "
        "[--grid-size=<grid size>] [partition_method=random / biasrandom / clustered / metis]\n"
        "[--v]\n"
        "\n"
        "Graph types and args:\n"
        "  market [<file>]\n"
        "    Reads a Matrix-Market coordinate-formatted graph of directed/undirected\n"
        "    edges from stdin (or from the optionally-specified file).\n"
        
        "  --device=<device_index>  Set GPU device for running the graph primitive.\n"
        "  --undirected If set then treat the graph as undirected.\n"
        "  --instrumented If set then kernels keep track of queue-search_depth\n"
        "  and barrier duty (a relative indicator of load imbalance.)\n"
        "  --src Begins BFS from the vertex <source index>. If set as randomize\n"
        "  then will begin with a random source vertex.\n"
        "  If set as largestdegree then will begin with the node which has\n"
        "  largest degree.\n"
        "  --quick If set will skip the CPU validation code.\n"
        "  --mark-pred If set then keep not only label info but also predecessor info.\n"
        "  --queue-sizing Allocates a frontier queue sized at (graph-edges * <scale factor>).\n"
        "  Default is 1.0\n"
        );
 }

 /**
  * @brief Displays the BFS result (i.e., distance from source)
  *
  * @param[in] source_path Search depth from the source for each node.
  * @param[in] preds Predecessor node id for each node.
  * @param[in] nodes Number of nodes in the graph.
  * @param[in] MARK_PREDECESSORS Whether to show predecessor of each node.
  */
template<typename VertexId, typename SizeT>
void DisplaySolution(VertexId *source_path, VertexId *preds, SizeT nodes, bool MARK_PREDECESSORS, bool ENABLE_IDEMPOTENCE)
{
    if (nodes > 40)
        nodes = 40;
    printf("[");
    for (VertexId i = 0; i < nodes; ++i) {
        PrintValue(i);
        printf(":");
        PrintValue(source_path[i]);
        if (MARK_PREDECESSORS && !ENABLE_IDEMPOTENCE) {
            printf(",");
            PrintValue(preds[i]);
        }
        printf(" ");
    }
    printf("]\n");
}

 /**
  * Performance/Evaluation statistics
  */ 

struct Stats {
    const char *name;
    Statistic rate;
    Statistic search_depth;
    Statistic redundant_work;
    Statistic duty;

    Stats() : name(NULL), rate(), search_depth(), redundant_work(), duty() {}
    Stats(const char *name) : name(name), rate(), search_depth(), redundant_work(), duty() {}
};

/**
 * @brief Displays timing and correctness statistics
 *
 * @tparam MARK_PREDECESSORS
 * @tparam VertexId
 * @tparam Value
 * @tparam SizeT
 * 
 * @param[in] stats Reference to the Stats object defined in RunTests
 * @param[in] src Source node where BFS starts
 * @param[in] h_labels Host-side vector stores computed labels for validation
 * @param[in] graph Reference to the CSR graph we process on
 * @param[in] elapsed Total elapsed kernel running time
 * @param[in] search_depth Maximum search depth of the BFS algorithm
 * @param[in] total_queued Total element queued in BFS kernel running process
 * @param[in] avg_duty Average duty of the BFS kernels
 */
template<
    bool MARK_PREDECESSORS,
    typename VertexId,
    typename Value,
    typename SizeT>
void DisplayStats(
    Stats               &stats,
    VertexId            src,
    VertexId            *h_labels,
    const Csr<VertexId, Value, SizeT> &graph,
    double              elapsed,
    VertexId            search_depth,
    long long           total_queued,
    double              avg_duty)
{
    // Compute nodes and edges visited
    SizeT edges_visited = 0;
    SizeT nodes_visited = 0;
    for (VertexId i = 0; i < graph.nodes; ++i) {
        if (h_labels[i] < util::MaxValue<VertexId>() && h_labels[i]!=-1) {
            ++nodes_visited;
            edges_visited += graph.row_offsets[i+1] - graph.row_offsets[i];
        }
    }

    double redundant_work = 0.0;
    if (total_queued > 0) {
        redundant_work = ((double) total_queued - edges_visited) / edges_visited;        // measure duplicate edges put through queue
    }
    redundant_work *= 100;

    // Display test name
    printf("[%s] finished. ", stats.name);

    // Display statistics
    if (nodes_visited < 5) {
        printf("Fewer than 5 vertices visited.\n");
    } else {
        // Display the specific sample statistics
        double m_teps = (double) edges_visited / (elapsed * 1000.0);
        printf(" elapsed: %.3f ms, rate: %.3f MiEdges/s", elapsed, m_teps);
        if (search_depth != 0) printf(", search_depth: %lld", (long long) search_depth);
        if (avg_duty != 0) {
            printf("\n avg CTA duty: %.2f%%", avg_duty * 100);
        }
        printf("\n src: %lld, nodes_visited: %lld, edges visited: %lld",
            (long long) src, (long long) nodes_visited, (long long) edges_visited);
        if (total_queued > 0) {
            printf(", total queued: %lld", total_queued);
        }
        if (redundant_work > 0) {
            printf(", redundant work: %.2f%%", redundant_work);
        }
        printf("\n");
    }
    
}


/******************************************************************************
 * BFS Testing Routines
 *****************************************************************************/

 /**
  * @brief A simple CPU-based reference BFS ranking implementation.
  *
  * @tparam VertexId
  * @tparam Value
  * @tparam SizeT
  *
  * @param[in] graph Reference to the CSR graph we process on
  * @param[in] source_path Host-side vector to store CPU computed labels for each node
  * @param[in] src Source node where BFS starts
  */
template<
    typename VertexId,
    typename Value,
    typename SizeT,
    bool MARK_PREDECESSORS,
    bool ENABLE_IDEMPOTENCE>
void SimpleReferenceBfs(
    const Csr<VertexId, Value, SizeT>       &graph,
    VertexId                                *source_path,
    VertexId                                *predecessor,
    VertexId                                src)
{
    //initialize distances
    for (VertexId i = 0; i < graph.nodes; ++i) {
        source_path[i] = ENABLE_IDEMPOTENCE? -1: util::MaxValue<VertexId>()-1;
        //source_path[i] = -1;
        if (MARK_PREDECESSORS)
            predecessor[i] = -1;
    }
    source_path[src] = 0;
    VertexId search_depth = 0;

    // Initialize queue for managing previously-discovered nodes
    std::deque<VertexId> frontier;
    frontier.push_back(src);

    //
    //Perform BFS
    //

    CpuTimer cpu_timer;
    cpu_timer.Start();
    while (!frontier.empty()) {
        
        // Dequeue node from frontier
        VertexId dequeued_node = frontier.front();
        frontier.pop_front();
        VertexId neighbor_dist = source_path[dequeued_node] + 1;

        // Locate adjacency list
        int edges_begin = graph.row_offsets[dequeued_node];
        int edges_end = graph.row_offsets[dequeued_node + 1];

        for (int edge = edges_begin; edge < edges_end; ++edge) {
            //Lookup neighbor and enqueue if undiscovered
            VertexId neighbor = graph.column_indices[edge];
            if (source_path[neighbor] > neighbor_dist || source_path[neighbor] == -1) {
                source_path[neighbor] = neighbor_dist;
                if (MARK_PREDECESSORS)
                    predecessor[neighbor] = dequeued_node;
                if (search_depth < neighbor_dist) {
                    search_depth = neighbor_dist;
                }
                frontier.push_back(neighbor);
            }
        }
    }

    if (MARK_PREDECESSORS)
        predecessor[src] = -1;

    cpu_timer.Stop();
    float elapsed = cpu_timer.ElapsedMillis();
    search_depth++;

    printf("CPU BFS finished in %lf msec. Search depth is:%d\n", elapsed, search_depth);
}

/**
 * @brief Run BFS tests
 *
 * @tparam VertexId
 * @tparam Value
 * @tparam SizeT
 * @tparam INSTRUMENT
 * @tparam MARK_PREDECESSORS
 *
 * @param[in] graph Reference to the CSR graph we process on
 * @param[in] src Source node where BFS starts
 * @param[in] max_grid_size Maximum CTA occupancy
 * @param[in] num_gpus Number of GPUs
 * @param[in] max_queue_sizing Scaling factor used in edge mapping
 *
 */
template <
    typename    VertexId,
    typename    Value,
    typename    SizeT,
    bool        INSTRUMENT,
    bool        MARK_PREDECESSORS,
    bool        ENABLE_IDEMPOTENCE>
void RunTests(
    Csr<VertexId, Value, SizeT> 
                 &graph,
    VertexId     src,
    int          max_grid_size,
    int          num_gpus,
    double       max_queue_sizing,
    double       max_in_sizing,
    ContextPtr   *context,
    std::string  partition_method,
    int          *gpu_idx,
    hipStream_t *streams,
    bool         size_check = true)
{
    size_t *org_size=new size_t[num_gpus];

    typedef BFSProblem<
        VertexId,
        SizeT,
        Value,
        MARK_PREDECESSORS,
        ENABLE_IDEMPOTENCE,
        (MARK_PREDECESSORS && ENABLE_IDEMPOTENCE)> Problem; // does not use double buffer

    // Allocate host-side label array (for both reference and gpu-computed results)
    VertexId    *reference_labels       = new VertexId[graph.nodes];
    VertexId    *reference_preds        = new VertexId[graph.nodes];
    VertexId    *h_labels               = new VertexId[graph.nodes];
    VertexId    *reference_check_label  = (g_quick) ? NULL : reference_labels;
    VertexId    *reference_check_preds  = NULL;
    VertexId    *h_preds                = NULL;
    if (MARK_PREDECESSORS) {
        h_preds = new VertexId[graph.nodes];
        if (!g_quick) {
              reference_check_preds = reference_preds;
        }            
    } 

    for (int gpu=0;gpu<num_gpus;gpu++)
    {
        size_t dummy;
        hipSetDevice(gpu_idx[gpu]);
        hipMemGetInfo(&(org_size[gpu]),&dummy);
    }
    // Allocate BFS enactor map
    BFSEnactor<Problem, INSTRUMENT>* bfs_enactor
         = new BFSEnactor<Problem, INSTRUMENT>(g_verbose, num_gpus, gpu_idx);
            
    // Allocate problem on GPU
    Problem *csr_problem = new Problem;
    util::GRError(csr_problem->Init(
        g_stream_from_host,
        graph,
        NULL,
        num_gpus,
        gpu_idx,
        partition_method,
        streams,
        max_queue_sizing,
        max_in_sizing), "Problem BFS Initialization Failed", __FILE__, __LINE__);
    util::GRError(bfs_enactor->Init (context, csr_problem, max_grid_size, size_check), "BFS Enactor init failed", __FILE__, __LINE__);
    //
    // Compute reference CPU BFS solution for source-distance
    //
    if (reference_check_label != NULL)
    {
        printf("compute ref value\n");
        SimpleReferenceBfs<VertexId, Value, SizeT, MARK_PREDECESSORS, ENABLE_IDEMPOTENCE>(
            graph,
            reference_check_label,
            reference_check_preds,
            src);
        printf("\n");
    }

    Stats     *stats       = new Stats("GPU BFS");
    long long total_queued = 0;
    VertexId  search_depth = 0;
    double    avg_duty     = 0.0; 

    // Perform BFS
    CpuTimer cpu_timer;

    util::GRError(csr_problem->Reset(src, bfs_enactor->GetFrontierType(), max_queue_sizing), "BFS Problem Data Reset Failed", __FILE__, __LINE__);
    util::GRError(bfs_enactor->Reset(), "BFS Enactor Reset failed", __FILE__, __LINE__);

    util::GRError("Error before Enact", __FILE__, __LINE__);
    cpu_timer.Start();
    util::GRError(bfs_enactor->Enact(src), "BFS Problem Enact Failed", __FILE__, __LINE__);
    cpu_timer.Stop();

    bfs_enactor->GetStatistics(total_queued, search_depth, avg_duty);

    float elapsed = cpu_timer.ElapsedMillis();

    // Copy out results
    util::GRError(csr_problem->Extract(h_labels, h_preds), "BFS Problem Data Extraction Failed", __FILE__, __LINE__);

    // Verify the result
    if (reference_check_label != NULL) {
        if (!ENABLE_IDEMPOTENCE) {
            printf("Label Validity: ");
            int error_num = CompareResults(h_labels, reference_check_label, graph.nodes, true);
            if (error_num > 0)
                printf("%d errors occurred.\n", error_num);
        } else {
            if (!MARK_PREDECESSORS) {
                printf("Label Validity: ");
                int error_num = CompareResults(h_labels, reference_check_label, graph.nodes, true);
                if (error_num > 0)
                    printf("%d errors occurred.\n", error_num);
            }
        }
    }
    printf("\nFirst 40 labels of the GPU result."); 
    // Display Solution
    DisplaySolution(h_labels, h_preds, graph.nodes, MARK_PREDECESSORS, ENABLE_IDEMPOTENCE);

    DisplayStats<MARK_PREDECESSORS>(
        *stats,
        src,
        h_labels,
        graph,
        elapsed,
        search_depth,
        total_queued,
        avg_duty);

    
    printf("\n\tMemory Usage(B)\t");
    for (int gpu=0;gpu<num_gpus;gpu++) 
    if (num_gpus>1) printf(" #keys%d\t #ins%d\t",gpu,gpu);
    else printf(" #keys%d", gpu);
    if (num_gpus>1) printf(" #keys%d",num_gpus);
    printf("\n");
    double max_key_sizing=0, max_in_sizing_=0;
    for (int gpu=0;gpu<num_gpus;gpu++)
    {
        size_t gpu_free,dummy;
        hipSetDevice(gpu_idx[gpu]);
        hipMemGetInfo(&gpu_free,&dummy);
        printf("GPU_%d\t %ld",gpu_idx[gpu],org_size[gpu]-gpu_free);
        for (int i=0;i<num_gpus;i++)
        {
            SizeT x=csr_problem->graph_slices[gpu]->frontier_queues[i].keys[0].GetSize();
            printf("\t %d", x);
            double factor = 1.0*x/(num_gpus>1?csr_problem->graph_slices[gpu]->in_counter[i]:csr_problem->graph_slices[gpu]->nodes);
            if (factor > max_key_sizing) max_key_sizing=factor;
            if (num_gpus>1) 
            {
                x=csr_problem->data_slices[gpu][0].keys_in[0][i].GetSize();
                printf("\t %d", x);
                factor = 1.0*x/csr_problem->graph_slices[gpu]->in_counter[i];
                if (factor > max_in_sizing_) max_in_sizing_=factor;
            }
        }
        if (num_gpus>1) printf("\t %d",csr_problem->graph_slices[gpu]->frontier_queues[num_gpus].keys[0].GetSize());
        printf("\n");
    }
    printf("\t key_sizing =\t %lf", max_key_sizing);
    if (num_gpus>1) printf("\t in_sizing =\t %lf", max_in_sizing_);
    printf("\n");

    // Cleanup
    if (org_size        ) {delete[] org_size        ; org_size         = NULL;}
    if (stats           ) {delete   stats           ; stats            = NULL;}
    if (bfs_enactor     ) {delete   bfs_enactor     ; bfs_enactor      = NULL;}
    if (csr_problem     ) {delete   csr_problem     ; csr_problem      = NULL;}
    if (reference_labels) {delete[] reference_labels; reference_labels = NULL;}
    if (h_labels        ) {delete[] h_labels        ; h_labels         = NULL;}
    if (h_preds         ) {delete[] h_preds         ; h_preds          = NULL;}

    //hipDeviceSynchronize();
}

/**
 * @brief RunTests entry
 *
 * @tparam VertexId
 * @tparam Value
 * @tparam SizeT
 *
 * @param[in] graph Reference to the CSR graph we process on
 * @param[in] args Reference to the command line arguments
 */
template <
    typename VertexId,
    typename Value,
    typename SizeT>
void RunTests(
    Csr<VertexId, Value, SizeT> &graph,
    CommandLineArgs             &args,
    int                         num_gpus,
    ContextPtr                  *context,
    int                         *gpu_idx,
    hipStream_t                *streams)
{
    VertexId            src                 = -1;           // Use whatever the specified graph-type's default is
    std::string         src_str;
    bool                instrumented        = false;        // Whether or not to collect instrumentation from kernels
    bool                mark_pred           = false;        // Whether or not to mark src-distance vs. parent vertices
    bool                idempotence         = false;        // Whether or not to enable idempotence operation
    int                 max_grid_size       = 0;            // maximum grid size (0: leave it up to the enactor)
    //int                 num_gpus            = 1;            // Number of GPUs for multi-gpu enactor to use
    double              max_queue_sizing    = 1.0;          // Maximum size scaling factor for work queues (e.g., 1.0 creates n and m-element vertex and edge frontiers).
    double              max_in_sizing       = 1.0;
    std::string         partition_method    = "random";
    bool                disable_size_check  = false;
    //int*                gpu_idx             = NULL;

    instrumented = args.CheckCmdLineFlag("instrumented");
    disable_size_check = args.CheckCmdLineFlag("disable-size-check");
    args.GetCmdLineArgument("src", src_str);
    if (src_str.empty()) {
        src = 0;
    } else if (src_str.compare("randomize") == 0) {
        src = graphio::RandomNode(graph.nodes);
    } else if (src_str.compare("largestdegree") == 0) {
        int temp;
        src = graph.GetNodeWithHighestDegree(temp);
    } else {
        args.GetCmdLineArgument("src", src);
    }
    printf("src = %d\n",src);
    printf("size_check = %s\n", disable_size_check?"false":"true");
    //printf("Display neighbor list of src:\n");
    //graph.DisplayNeighborList(src);

    g_verbose   = args.CheckCmdLineFlag("v");
    g_quick     = args.CheckCmdLineFlag("quick");
    mark_pred   = args.CheckCmdLineFlag("mark-pred");
    idempotence = args.CheckCmdLineFlag("idempotence");
    args.GetCmdLineArgument("queue-sizing", max_queue_sizing);
    args.GetCmdLineArgument("in-sizing", max_in_sizing);
    args.GetCmdLineArgument("grid-size",max_grid_size);
    if (args.CheckCmdLineFlag  ("partition_method")) 
        args.GetCmdLineArgument("partition_method",partition_method);
    //printf("partition_method0=%s\n",partition_method.c_str());
    if (instrumented) {
        if (mark_pred) {
            if (idempotence) {
                RunTests<VertexId, Value, SizeT, true, true, true>(
                        graph,
                        src,
                        max_grid_size,
                        num_gpus,
                        max_queue_sizing,
                        max_in_sizing,
                        context,
                        partition_method,
                        gpu_idx,
                        streams,
                        !disable_size_check);
            } else {
                RunTests<VertexId, Value, SizeT, true, true, false>(
                        graph,
                        src,
                        max_grid_size,
                        num_gpus,
                        max_queue_sizing,
                        max_in_sizing,
                        context,
                        partition_method,
                        gpu_idx,
                        streams,
                        !disable_size_check);
            }
        } else {
            if (idempotence) {
                RunTests<VertexId, Value, SizeT, true, false, true>(
                        graph,
                        src,
                        max_grid_size,
                        num_gpus,
                        max_queue_sizing,
                        max_in_sizing,
                        context,
                        partition_method,
                        gpu_idx,
                        streams,
                        !disable_size_check);
            } else {
                RunTests<VertexId, Value, SizeT, true, false, false>(
                        graph,
                        src,
                        max_grid_size,
                        num_gpus,
                        max_queue_sizing,
                        max_in_sizing,
                        context,
                        partition_method,
                        gpu_idx,
                        streams,
                        !disable_size_check);
            }
        }
    } else {
        if (mark_pred) {
            if (idempotence) {
                RunTests<VertexId, Value, SizeT, false, true, true>(
                        graph,
                        src,
                        max_grid_size,
                        num_gpus,
                        max_queue_sizing,
                        max_in_sizing,
                        context,
                        partition_method,
                        gpu_idx,
                        streams,
                        !disable_size_check);
            } else {
                RunTests<VertexId, Value, SizeT, false, true, false>(
                        graph,
                        src,
                        max_grid_size,
                        num_gpus,
                        max_queue_sizing,
                        max_in_sizing,
                        context,
                        partition_method,
                        gpu_idx,
                        streams,
                        !disable_size_check);
            }
        } else {
            if (idempotence) {
                RunTests<VertexId, Value, SizeT, false, false, true>(
                        graph,
                        src,
                        max_grid_size,
                        num_gpus,
                        max_queue_sizing,
                        max_in_sizing,
                        context,
                        partition_method,
                        gpu_idx,
                        streams,
                        !disable_size_check);
            } else {
                RunTests<VertexId, Value, SizeT, false, false, false>(
                        graph,
                        src,
                        max_grid_size,
                        num_gpus,
                        max_queue_sizing,
                        max_in_sizing,
                        context,
                        partition_method,
                        gpu_idx,
                        streams,
                        !disable_size_check);
            }
        }
    }

}



/******************************************************************************
* Main
******************************************************************************/

int cpp_main( int argc, char** argv)
{
    CommandLineArgs args(argc, argv);
    int          num_gpus = 0;
    int          *gpu_idx = NULL;
    ContextPtr   *context = NULL;
    hipStream_t *streams = NULL;

    if ((argc < 2) || (args.CheckCmdLineFlag("help"))) {
        Usage();
        return 1;
    }

    if (args.CheckCmdLineFlag  ("device"))
    {   
        std::vector<int> gpus;
        args.GetCmdLineArguments<int>("device",gpus);
        num_gpus   = gpus.size();
        gpu_idx    = new int[num_gpus];
        for (int i=0;i<num_gpus;i++) 
            gpu_idx[i] = gpus[i];
    } else {
        num_gpus   = 1;
        gpu_idx    = new int[num_gpus];
        gpu_idx[0] = 0;
    }
    streams  = new hipStream_t[num_gpus * num_gpus *2];
    context  = new ContextPtr  [num_gpus * num_gpus];
    printf("Using %d gpus: ", num_gpus);
    for (int gpu=0;gpu<num_gpus;gpu++) 
    {
        printf(" %d ", gpu_idx[gpu]);
        util::SetDevice(gpu_idx[gpu]);
        for (int i=0;i<num_gpus*2;i++)
        {
            int _i=gpu*num_gpus*2+i;
            util::GRError(hipStreamCreate(&streams[_i]), "hipStreamCreate fialed.",__FILE__,__LINE__);
            if (i<num_gpus) context[gpu*num_gpus+i] = mgpu::CreateCudaDeviceAttachStream(gpu_idx[gpu],streams[_i]);
        }
    }
    printf("\n"); fflush(stdout);
    
    /*for (int gpu=0;gpu<num_gpus;gpu++)
    {    
	util::SetDevice(gpu_idx[gpu]);
        util::Array1D<int,int> arr;
        arr.Init(1,util::HOST | util::DEVICE, true, hipHostMallocMapped | hipHostMallocPortable);
	for (int i=0;i<num_gpus;i++)
	{    
	    util::cpu_mt::PrintMessage("check point",gpu,i);
	    int _i=gpu*num_gpus+i;
            arr[0]=0;
            util::MemsetKernel<<<1,1,0,streams[_i]>>>(arr.GetPointer(util::DEVICE), 0, 1);
            hipStreamSynchronize(streams[_i]);
            util::GRError("MemsetKernel failed.", __FILE__, __LINE__);
	}
        arr.Release(); 
    } */   

    // Parse graph-contruction params
    g_undirected = args.CheckCmdLineFlag("undirected");

    std::string graph_type = argv[1];
    int flags = args.ParsedArgc();
    int graph_args = argc - flags - 1;

    if (graph_args < 1) {
        Usage();
        return 1;
    }

    //
    // Construct graph and perform search(es)
    //

    if (graph_type == "market") {

        // Matrix-market coordinate-formatted graph file

        typedef int VertexId;                   // Use as the node identifier type
        typedef int Value;                      // Use as the value type
        typedef int SizeT;                      // Use as the graph size type
        Csr<VertexId, Value, SizeT> csr(false); // default value for stream_from_host is false

        if (graph_args < 1) { Usage(); return 1; }
        char *market_filename = (graph_args == 2) ? argv[2] : NULL;
        if (graphio::BuildMarketGraph<false>(
            market_filename, 
            csr, 
            g_undirected,
            false) != 0) // no inverse graph
        {
            return 1;
        }

        csr.PrintHistogram();

        // Run tests
        RunTests(csr, args, num_gpus, context, gpu_idx, streams);

    } else {

        // Unknown graph type
        fprintf(stderr, "Unspecified graph type\n");
        return 1;

    }
    return 0;
}

